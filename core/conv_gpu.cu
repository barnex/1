#include "conv_gpu.h"

#include <hipfft/hipfft.h>
#include <stdio.h>

typedef struct{
  hipfftHandle handle;
  
  float* source;
  float* transf;
  
  float* device;
  float* device2;
  
  int N0, N1, N2;
  int direction;
} cudaPlan;



void* fft_init_forward(int N0, int N1, int N2, float* source, float* transf){
    cudaPlan* plan = (cudaPlan*) malloc(sizeof(cudaPlan));
    
    hipfftPlan3d(&(plan->handle), N0, N1, N2, HIPFFT_R2C);
    plan->source = source;
    plan->transf = transf;
    hipMalloc((void**)&plan->device, (N0*N1*(N2+2)) * sizeof(float));
    hipMalloc((void**)&plan->device2, (N0*N1*(N2+2)) * sizeof(float));
    plan->N0 = N0;
    plan->N1 = N1;
    plan->N2 = N2;
    plan->direction = HIPFFT_FORWARD;
    printf("fft_init_forward\t(%d, %d, %d):\t%p\n", N0, N1, N2, plan);
    return plan;
}


void* fft_init_backward(int N0, int N1, int N2, float* transf, float* source){
    cudaPlan* plan = (cudaPlan*) malloc(sizeof(cudaPlan));
    
    hipfftPlan3d(&(plan->handle), N0, N1, N2, HIPFFT_C2R);
    plan->source = source;
    plan->transf = transf;
    hipMalloc((void**)&plan->device, (N0*N1*(N2+2)) * sizeof(float));
    hipMalloc((void**)&plan->device2, (N0*N1*(N2+2)) * sizeof(float));
    plan->N0 = N0;
    plan->N1 = N1;
    plan->N2 = N2;
    plan->direction = HIPFFT_BACKWARD;
    printf("fft_init_backward\t(%d, %d, %d):\t%p\n", N0, N1, N2, plan);
    return plan;
}


void fft_execute(void* plan_ptr){
  
  printf("fft_execute():\t%p\n", plan_ptr);
  
  cudaPlan* plan = (cudaPlan*)plan_ptr;
  int N0 = plan->N0, N1 = plan->N1, N2 = plan->N2;
  //int N = plan->N0 * plan->N1 * plan->N2;
  printf("%d x %d x %d\n", N0, N1, N2);
  
  ///////////////////////////////////////////// forward ///////////////////////////////////////////
  
  if(plan->direction == HIPFFT_FORWARD){
    printf("fft_execute() [forward]:\t%p\n", plan_ptr);
    
    //printf("**ORIGINAL DATA:\n");
    //format_tensor(as_tensor(plan->source, 3, N0, N1, N2), stdout);
    
    printf("memcpy: %d\n", hipMemcpy(plan->device, plan->source, (N0*N1*N2) * sizeof(float), hipMemcpyHostToDevice));
    
    // r2c transform
    printf("r2c: %d\n",hipfftExecR2C(plan->handle, (hipfftReal*)plan->device, (hipfftComplex*)plan->device2));
    
    // copy everything back
    hipMemcpy(plan->transf, plan->device2, N0*N1*(N2+2) * sizeof(float), hipMemcpyDeviceToHost);
    
    //printf("**TRANSFORMED:\n");
    //format_tensor(as_tensor(plan->transf, 3, N0, N1, N2+2), stdout);
  }
  
  ////////////////////////////////////////// backward ///////////////////////////////////////////////
  
  else if (plan->direction == HIPFFT_BACKWARD){
    printf("fft_execute() [backward]:\t%p\n", plan_ptr);
    
    //printf("**BACKTRANSF INPUT (HALF):\n"); 
    //format_tensor(as_tensor(plan->transf, 3, N0, N1, N2+2), stdout); 
    
    printf("memcpy: %d\n", hipMemcpy(plan->device, plan->transf, N0*N1*(N2+2) * sizeof(float), hipMemcpyHostToDevice));
    
    hipfftExecC2R(plan->handle, (hipfftComplex*)plan->device, (hipfftReal*)plan->device2);
    
    printf("memcpy: %d\n",hipMemcpy(plan->source, plan->device2, (N0*N1*N2) * sizeof(float), hipMemcpyDeviceToHost));
    
    //printf("**BACKTRANSF (HALF):\n"); 
    //format_tensor(as_tensor(plan->source, 3, N0, N1, N2), stdout);

  }
  /////////////////////////////////// not backward nor forward ///////////////////////////
  
  else{
    printf("fft_execute() [illegal plan]:\t%p", plan_ptr);
    exit(3);
  }
}










void fft_init(void){
}


void fft_finalize(void){
}

float* fft_malloc(int N0, int N1, int N2){
    return 0;
}


void fft_free(void* data){
}

void fft_destroy_plan(void* plan){
}
