#include "hip/hip_runtime.h"
#include "gpu_micromag3d_kernel.h"

#ifdef __cplusplus
extern "C" {
#endif


tensor *gpu_micromag3d_kernel(param* p){
	
  // check input + allocate tensor on device ______________________________________________________
    check_param(p);
    int kernelStorageN = p->kernelSize[X] * p->kernelSize[Y] * gpu_pad_to_stride(p->kernelSize[Z]+2);
    tensor *dev_kernel;
    if (p->size[X]==0)
      dev_kernel = as_tensor(new_gpu_array(4*kernelStorageN/2), 2, 4, kernelStorageN/2);  // only real parts!!
    else
      dev_kernel = as_tensor(new_gpu_array(6*kernelStorageN/2), 2, 6, kernelStorageN/2);  // only real parts!!
	// ______________________________________________________________________________________________


	// initialization Gauss quadrature points for integrations + copy to gpu ________________________
		float *dev_qd_W_10 = new_gpu_array(10);
		float *dev_qd_P_10 = new_gpu_array(3*10);
		initialize_Gauss_quadrature_on_gpu(dev_qd_W_10, dev_qd_P_10, p->cellSize);
	// ______________________________________________________________________________________________
	
	
	// Plan initialization for FFTs Greens kernel elements __________________________________________
		int* zero_pad_kernel = (int*)calloc(3, sizeof(int));
		zero_pad_kernel[X] = zero_pad_kernel[Y] = zero_pad_kernel[Z] = 0; 
		gpu_plan3d_real_input* kernel_plan = new_gpu_plan3d_real_input(p->kernelSize[X], p->kernelSize[Y], p->kernelSize[Z], zero_pad_kernel);
	// ______________________________________________________________________________________________


	// Initialize the kernel ________________________________________________________________________		
		gpu_init_and_FFT_Greens_kernel_elements(dev_kernel, p->kernelSize, p->cellSize, p->demagPeriodic, dev_qd_P_10, dev_qd_W_10, kernel_plan);
	// ______________________________________________________________________________________________	
	
	return (dev_kernel);
}

/// @todo argument defining which Greens function should be added
/// remark: number of FD cells in a dimension can not be odd if no zero padding!!
void gpu_init_and_FFT_Greens_kernel_elements(tensor *dev_kernel, int *demagKernelSize, float *FD_cell_size, int *repetition, float *dev_qd_P_10, float *dev_qd_W_10, gpu_plan3d_real_input* kernel_plan){

  
 	int kernelStorageN = 2*dev_kernel->size[1];				// size of kernel component in real + i*complex format
	float *dev_temp = new_gpu_array(kernelStorageN);		// temp tensor on device for storage of each component in real + i*complex format
 	
	// Define gpugrids and blocks ___________________________________________________________________
    dim3 gridsize1(demagKernelSize[X]/2, demagKernelSize[Y]/2, 1);  ///@todo generalize!
    if (demagKernelSize[X]==1)  //overwrites last line if simulation with thickness = 1 FD cell
      dim3 gridsize1(1, demagKernelSize[Y]/2, 1);	                  ///@todo generalize!

    dim3 blocksize1(demagKernelSize[Z]/2, 1, 1);				         ///@todo aan te passen!!  GPU_STRIDE_FLOAT
		gpu_checkconf(gridsize1, blocksize1);
		int gridsize2, blocksize2;
		make1dconf(kernelStorageN/2, &gridsize2, &blocksize2);
	// ______________________________________________________________________________________________
	

	// Main function operations _____________________________________________________________________
		int rank0 = 0;																			// defines the first rank of the Greens kernel [xx, xy, xz, yy, yz, zz]
    int max_co = (demagKernelSize[X]==1)? 2:3;
    for (int co1=0; co1<max_co; co1++){											// for a Greens kernel component [co1,co2]:
			for (int co2=co1; co2<max_co; co2++){
					// Put all elements in 'dev_temp' to zero.
				gpu_zero(dev_temp, kernelStorageN);		 
				hipDeviceSynchronize();
					// Fill in the elements.
				_gpu_init_Greens_kernel_elements<<<gridsize1, blocksize1>>>(dev_temp, demagKernelSize[X], demagKernelSize[Y], demagKernelSize[Z], co1, co2, FD_cell_size[X], FD_cell_size[Y], FD_cell_size[Z], repetition[X], repetition[Y], repetition[Z], dev_qd_P_10, dev_qd_W_10);
				hipDeviceSynchronize();
					// Fourier transform the kernel component.
				gpu_plan3d_real_input_forward(kernel_plan, dev_temp);
				hipDeviceSynchronize();
					// Copy the real parts to the corresponding place in the dev_kernel tensor.
				_gpu_extract_real_parts<<<gridsize2, blocksize2>>>(&dev_kernel->list[rank0*kernelStorageN/2], dev_temp, rank0, kernelStorageN/2);
				hipDeviceSynchronize();
				rank0++;																				// get ready for next component
			}
		}
	// ______________________________________________________________________________________________

	hipFree (dev_temp);
	
	return;
}



__global__ void _gpu_init_Greens_kernel_elements(float *dev_temp, int Nkernel_X, int Nkernel_Y, int Nkernel_Z, int co1, int co2, float FD_cell_size_X, float FD_cell_size_Y, float FD_cell_size_Z, int repetition_X, int repetition_Y, int repetition_Z, float *dev_qd_P_10, float *dev_qd_W_10){
   
	int i = blockIdx.x;
	int j = blockIdx.y;
	int k = threadIdx.x;

	int N2 = Nkernel_Z+2;     ///@todo: a gpu_pad_to_stride() function also executable on gpu should be used here
	int N12 = Nkernel_Y * N2;

		dev_temp[            i*N12 +             j*N2 +           k] = _gpu_get_Greens_element(Nkernel_X, Nkernel_Y, Nkernel_Z, co1, co2,  i,  j,  k, FD_cell_size_X, FD_cell_size_Y, FD_cell_size_Z, repetition_X, repetition_Y, repetition_Z, dev_qd_P_10, dev_qd_W_10);
	if (i>0)
		dev_temp[(Nkernel_X-i)*N12 +             j*N2 +           k] = _gpu_get_Greens_element(Nkernel_X, Nkernel_Y, Nkernel_Z, co1, co2, -i,  j,  k, FD_cell_size_X, FD_cell_size_Y, FD_cell_size_Z, repetition_X, repetition_Y, repetition_Z, dev_qd_P_10, dev_qd_W_10);
	if (j>0)
		dev_temp[            i*N12 + (Nkernel_Y-j)*N2 +           k] = _gpu_get_Greens_element(Nkernel_X, Nkernel_Y, Nkernel_Z, co1, co2,  i, -j,  k, FD_cell_size_X, FD_cell_size_Y, FD_cell_size_Z, repetition_X, repetition_Y, repetition_Z, dev_qd_P_10, dev_qd_W_10);
	if (k>0) 
		dev_temp[            i*N12 +             j*N2 + Nkernel_Z-k] = _gpu_get_Greens_element(Nkernel_X, Nkernel_Y, Nkernel_Z, co1, co2,  i,  j, -k, FD_cell_size_X, FD_cell_size_Y, FD_cell_size_Z, repetition_X, repetition_Y, repetition_Z, dev_qd_P_10, dev_qd_W_10);
	if (i>0 && j>0)
		dev_temp[(Nkernel_X-i)*N12 + (Nkernel_Y-j)*N2 +           k] = _gpu_get_Greens_element(Nkernel_X, Nkernel_Y, Nkernel_Z, co1, co2, -i, -j,  k, FD_cell_size_X, FD_cell_size_Y, FD_cell_size_Z, repetition_X, repetition_Y, repetition_Z, dev_qd_P_10, dev_qd_W_10);
	if (i>0 && k>0) 
		dev_temp[(Nkernel_X-i)*N12 +             j*N2 + Nkernel_Z-k] = _gpu_get_Greens_element(Nkernel_X, Nkernel_Y, Nkernel_Z, co1, co2, -i,  j, -k, FD_cell_size_X, FD_cell_size_Y, FD_cell_size_Z, repetition_X, repetition_Y, repetition_Z, dev_qd_P_10, dev_qd_W_10);
	if (j>0 && k>0) 
		dev_temp[            i*N12 + (Nkernel_Y-j)*N2 + Nkernel_Z-k] = _gpu_get_Greens_element(Nkernel_X, Nkernel_Y, Nkernel_Z, co1, co2,  i, -j, -k, FD_cell_size_X, FD_cell_size_Y, FD_cell_size_Z, repetition_X, repetition_Y, repetition_Z, dev_qd_P_10, dev_qd_W_10);
	if (i>0 && j>0 && k>0) 
		dev_temp[(Nkernel_X-i)*N12 + (Nkernel_Y-j)*N2 + Nkernel_Z-k] = _gpu_get_Greens_element(Nkernel_X, Nkernel_Y, Nkernel_Z, co1, co2, -i, -j, -k, FD_cell_size_X, FD_cell_size_Y, FD_cell_size_Z, repetition_X, repetition_Y, repetition_Z, dev_qd_P_10, dev_qd_W_10);

	return;
}



__device__ float _gpu_get_Greens_element(int Nkernel_X, int Nkernel_Y, int Nkernel_Z, int co1, int co2, int a, int b, int c, float FD_cell_size_X, float FD_cell_size_Y, float FD_cell_size_Z, int repetition_X, int repetition_Y, int repetition_Z, float *dev_qd_P_10, float *dev_qd_W_10){

	float result = 0.0f;
	float *dev_qd_P_10_X = &dev_qd_P_10[X];
	float *dev_qd_P_10_Y = &dev_qd_P_10[Y];
	float *dev_qd_P_10_Z = &dev_qd_P_10[Z];
	float dim_inverse = 1.0f/( (float) Nkernel_X*Nkernel_Y*Nkernel_Z  );
	
	// for elements in Kernel component gxx _________________________________________________________
		if (co1==0 && co2==0){
			for(int cnta=-repetition_X; cnta<=repetition_X; cnta++)
			for(int cntb=-repetition_Y; cntb<=repetition_Y; cntb++)
			for(int cntc=-repetition_Z; cntc<=repetition_Z; cntc++){

				int i = a + cnta*Nkernel_X/2;
				int j = b + cntb*Nkernel_Y/2;
				int k = c + cntc*Nkernel_Z/2;
				int r2_int = i*i+j*j+k*k;

				if (r2_int<400){
					float x1 = (i + 0.5f) * FD_cell_size_X;
					float x2 = (i - 0.5f) * FD_cell_size_X;
					for (int cnt2=0; cnt2<10; cnt2++){
						float y = j * FD_cell_size_Y + dev_qd_P_10_Y[cnt2];
						for (int cnt3=0; cnt3<10; cnt3++){
							float z = k * FD_cell_size_Z + dev_qd_P_10_Z[cnt3];
							result += FD_cell_size_Y * FD_cell_size_Z / 4.0f * dev_qd_W_10[cnt2] * dev_qd_W_10[cnt3] *
								( x1*__powf(x1*x1+y*y+z*z, -1.5f) - x2*__powf(x2*x2+y*y+z*z, -1.5f));
						}
					}
				}
				else{
					float r2 = (i*FD_cell_size_X)*(i*FD_cell_size_X) + (j*FD_cell_size_Y)*(j*FD_cell_size_Y) + (k*FD_cell_size_Z)*(k*FD_cell_size_Z);
					result += FD_cell_size_X * FD_cell_size_Y * FD_cell_size_Z *
										(1.0f/ __powf(r2,1.5f) - 3.0f* (i*FD_cell_size_X) * (i*FD_cell_size_X) * __powf(r2,-2.5f));
				}
			}
      result *= -1.0f/4.0f/3.14159265f;

			if (a== 1 && b== 0 && c== 0)	result -= 2.0f/FD_cell_size_X/FD_cell_size_X;						//exchange contribution
			if (a==-1 && b== 1 && c== 0)	result -= 2.0f/FD_cell_size_X/FD_cell_size_X;						//exchange contribution
			if (a== 0 && b== 0 && c== 0)	result -= 2.0f/FD_cell_size_Y/FD_cell_size_Y;						//exchange contribution
			if (a== 0 && b==-1 && c== 0)	result -= 2.0f/FD_cell_size_Y/FD_cell_size_Y;						//exchange contribution
			if (a== 0 && b== 0 && c== 1)	result -= 2.0f/FD_cell_size_Z/FD_cell_size_Z;						//exchange contribution
			if (a== 0 && b== 0 && c==-1)	result -= 2.0f/FD_cell_size_Z/FD_cell_size_Z;						//exchange contribution
		}
	// ______________________________________________________________________________________________


	// for elements in Kernel component gxy _________________________________________________________
		if (co1==0 && co2==1){
			for(int cnta=-repetition_X; cnta<=repetition_X; cnta++)
			for(int cntb=-repetition_Y; cntb<=repetition_Y; cntb++)
			for(int cntc=-repetition_Z; cntc<=repetition_Z; cntc++){

				int i = a + cnta*Nkernel_X/2;
				int j = b + cntb*Nkernel_Y/2;
				int k = c + cntc*Nkernel_Z/2;
				int r2_int = i*i+j*j+k*k;

				if (r2_int<400){
					float x1 = (i + 0.5f) * FD_cell_size_X;
					float x2 = (i - 0.5f) * FD_cell_size_X;
					for (int cnt2=0; cnt2<10; cnt2++){
						float y = j * FD_cell_size_Y + dev_qd_P_10_Y[cnt2];
						for (int cnt3=0; cnt3<10; cnt3++){
							float z = k * FD_cell_size_Z + dev_qd_P_10_Z[cnt3];
							result += FD_cell_size_Y * FD_cell_size_Z / 4.0f * dev_qd_W_10[cnt2] * dev_qd_W_10[cnt3] *
								( y*__powf(x1*x1+y*y+z*z, -1.5f) - y*__powf(x2*x2+y*y+z*z, -1.5f));
						}
					}
				}
				else{
					float r2 = (i*FD_cell_size_X)*(i*FD_cell_size_X) + (j*FD_cell_size_Y)*(j*FD_cell_size_Y) + (k*FD_cell_size_Z)*(k*FD_cell_size_Z);
					result += FD_cell_size_X * FD_cell_size_Y * FD_cell_size_Z * 
										(- 3.0f* (i*FD_cell_size_X) * (j*FD_cell_size_Y) * __powf(r2,-2.5f));
				}
			}
      result *= -1.0f/4.0f/3.14159265f;
		}
	// ______________________________________________________________________________________________


	// for elements in Kernel component gxz _________________________________________________________
		if (co1==0 && co2==2){
			for(int cnta=-repetition_X; cnta<=repetition_X; cnta++)
			for(int cntb=-repetition_Y; cntb<=repetition_Y; cntb++)
			for(int cntc=-repetition_Z; cntc<=repetition_Z; cntc++){

				int i = a + cnta*Nkernel_X/2;
				int j = b + cntb*Nkernel_Y/2;
				int k = c + cntc*Nkernel_Z/2;
				int r2_int = i*i+j*j+k*k;

				if (r2_int<400){
					float x1 = (i + 0.5f) * FD_cell_size_X;
					float x2 = (i - 0.5f) * FD_cell_size_X;
					for (int cnt2=0; cnt2<10; cnt2++){
						float y = j * FD_cell_size_Y + dev_qd_P_10_Y[cnt2];
						for (int cnt3=0; cnt3<10; cnt3++){
							float z = k * FD_cell_size_Z + dev_qd_P_10_Z[cnt3];
							result += FD_cell_size_Y * FD_cell_size_Z / 4.0f * dev_qd_W_10[cnt2] * dev_qd_W_10[cnt3] *
								( z*__powf(x1*x1+y*y+z*z, -1.5f) - z*__powf(x2*x2+y*y+z*z, -1.5f));
						}
					}
				}
				else{
					float r2 = (i*FD_cell_size_X)*(i*FD_cell_size_X) + (j*FD_cell_size_Y)*(j*FD_cell_size_Y) + (k*FD_cell_size_Z)*(k*FD_cell_size_Z);
					result += FD_cell_size_X * FD_cell_size_Y * FD_cell_size_Z * 
										(- 3.0f* (i*FD_cell_size_X) * (k*FD_cell_size_Y) * __powf(r2,-2.5f));
				}
			}
      result *= -1.0f/4.0f/3.14159265f;
		}
	// ______________________________________________________________________________________________


	// for elements in Kernel component gyy _________________________________________________________
		if (co1==1 && co2==1){
			for(int cnta=-repetition_X; cnta<=repetition_X; cnta++)
			for(int cntb=-repetition_Y; cntb<=repetition_Y; cntb++)
			for(int cntc=-repetition_Z; cntc<=repetition_Z; cntc++){

				int i = a + cnta*Nkernel_X/2;
				int j = b + cntb*Nkernel_Y/2;
				int k = c + cntc*Nkernel_Z/2;
				int r2_int = i*i+j*j+k*k;

				if (r2_int<400){
					float y1 = (j + 0.5f) * FD_cell_size_Y;
					float y2 = (j - 0.5f) * FD_cell_size_Y;
					for (int cnt1=0; cnt1<10; cnt1++){
						float x = i * FD_cell_size_X + dev_qd_P_10_X[cnt1];
						for (int cnt3=0; cnt3<10; cnt3++){
							float z = k * FD_cell_size_Z + dev_qd_P_10_Z[cnt3];
							result += FD_cell_size_X * FD_cell_size_Z / 4.0f * dev_qd_W_10[cnt1] * dev_qd_W_10[cnt3] *
								( y1*__powf(x*x+y1*y1+z*z, -1.5f) - y2*__powf(x*x+y2*y2+z*z, -1.5f));
						}
					}
				}
				else{
					float r2 = (i*FD_cell_size_X)*(i*FD_cell_size_X) + (j*FD_cell_size_Y)*(j*FD_cell_size_Y) + (k*FD_cell_size_Z)*(k*FD_cell_size_Z);
					result += FD_cell_size_X * FD_cell_size_Y * FD_cell_size_Z * 
										(1.0f/ __powf(r2,1.5f) - 3.0f* (j*FD_cell_size_Y) * (j*FD_cell_size_Y) * __powf(r2,-2.5f));
				}
			}
      result *= -1.0f/4.0f/3.14159265f;

			if (a== 1 && b== 0 && c== 0)	result -= 2.0f/FD_cell_size_X/FD_cell_size_X;						//exchange contribution
			if (a==-1 && b== 1 && c== 0)	result -= 2.0f/FD_cell_size_X/FD_cell_size_X;						//exchange contribution
			if (a== 0 && b== 0 && c== 0)	result -= 2.0f/FD_cell_size_Y/FD_cell_size_Y;						//exchange contribution
			if (a== 0 && b==-1 && c== 0)	result -= 2.0f/FD_cell_size_Y/FD_cell_size_Y;						//exchange contribution
			if (a== 0 && b== 0 && c== 1)	result -= 2.0f/FD_cell_size_Z/FD_cell_size_Z;						//exchange contribution
			if (a== 0 && b== 0 && c==-1)	result -= 2.0f/FD_cell_size_Z/FD_cell_size_Z;						//exchange contribution
		}
	// ______________________________________________________________________________________________


	// for elements in Kernel component gyz _________________________________________________________
		if (co1==1 && co2==2){
			for(int cnta=-repetition_X; cnta<=repetition_X; cnta++)
			for(int cntb=-repetition_Y; cntb<=repetition_Y; cntb++)
			for(int cntc=-repetition_Z; cntc<=repetition_Z; cntc++){

				int i = a + cnta*Nkernel_X/2;
				int j = b + cntb*Nkernel_Y/2;
				int k = c + cntc*Nkernel_Z/2;
				int r2_int = i*i+j*j+k*k;

				if (r2_int<400){
					float y1 = (j + 0.5f) * FD_cell_size_Y;
					float y2 = (j - 0.5f) * FD_cell_size_Y;
					for (int cnt1=0; cnt1<10; cnt1++){
						float x = i * FD_cell_size_X + dev_qd_P_10_X[cnt1];
						for (int cnt3=0; cnt3<10; cnt3++){
							float z = k * FD_cell_size_Z + dev_qd_P_10_Z[cnt3];
							result += FD_cell_size_X * FD_cell_size_Z / 4.0f * dev_qd_W_10[cnt1] * dev_qd_W_10[cnt3] *
								( z*__powf(x*x+y1*y1+z*z, -1.5f) - z*__powf(x*x+y2*y2+z*z, -1.5f));
						}
					}
				}
				else{
					float r2 = (i*FD_cell_size_X)*(i*FD_cell_size_X) + (j*FD_cell_size_Y)*(j*FD_cell_size_Y) + (k*FD_cell_size_Z)*(k*FD_cell_size_Z);
					result += FD_cell_size_X * FD_cell_size_Y * FD_cell_size_Z * 
										( - 3.0f* (j*FD_cell_size_Y) * (k*FD_cell_size_Z) * __powf(r2,-2.5f));
				}
			}
      result *= -1.0f/4.0f/3.14159265f;
		}
	// ______________________________________________________________________________________________


	// for elements in Kernel component gzz _________________________________________________________
		if (co1==2 && co2==2){
			for(int cnta=-repetition_X; cnta<=repetition_X; cnta++)
			for(int cntb=-repetition_Y; cntb<=repetition_Y; cntb++)
			for(int cntc=-repetition_Z; cntc<=repetition_Z; cntc++){

				int i = a + cnta*Nkernel_X/2;
				int j = b + cntb*Nkernel_Y/2;
				int k = c + cntc*Nkernel_Z/2;
				int r2_int = i*i+j*j+k*k;

				if (r2_int<400){
					float z1 = (k + 0.5f) * FD_cell_size_Z;
					float z2 = (k - 0.5f) * FD_cell_size_Z;
					for (int cnt1=0; cnt1<10; cnt1++){
						float x = i * FD_cell_size_X + dev_qd_P_10_X[cnt1];
						for (int cnt2=0; cnt2<10; cnt2++){
							float y = j * FD_cell_size_Y + dev_qd_P_10_Y[cnt2];
							result += FD_cell_size_X * FD_cell_size_Y / 4.0f * dev_qd_W_10[cnt1] * dev_qd_W_10[cnt2] *
								( z1*__powf(x*x+y*y+z1*z1, -1.5f) - z2*__powf(x*x+y*y+z2*z2, -1.5f));
						}
					}
				}
				else{
					float r2 = (i*FD_cell_size_X)*(i*FD_cell_size_X) + (j*FD_cell_size_Y)*(j*FD_cell_size_Y) + (k*FD_cell_size_Z)*(k*FD_cell_size_Z);
					result += FD_cell_size_X * FD_cell_size_Y * FD_cell_size_Z * 
										(1.0f/ __powf(r2,1.5f) - 3.0f* (k*FD_cell_size_Z) * (k*FD_cell_size_Z) * __powf(r2,-2.5f));
				}
			}
      result *= -1.0f/4.0f/3.14159265f;

			if (a== 1 && b== 0 && c== 0)	result -= 2.0f/FD_cell_size_X/FD_cell_size_X;						//exchange contribution
			if (a==-1 && b== 1 && c== 0)	result -= 2.0f/FD_cell_size_X/FD_cell_size_X;						//exchange contribution
			if (a== 0 && b== 0 && c== 0)	result -= 2.0f/FD_cell_size_Y/FD_cell_size_Y;						//exchange contribution
			if (a== 0 && b==-1 && c== 0)	result -= 2.0f/FD_cell_size_Y/FD_cell_size_Y;						//exchange contribution
			if (a== 0 && b== 0 && c== 1)	result -= 2.0f/FD_cell_size_Z/FD_cell_size_Z;						//exchange contribution
			if (a== 0 && b== 0 && c==-1)	result -= 2.0f/FD_cell_size_Z/FD_cell_size_Z;						//exchange contribution
		}
	// ______________________________________________________________________________________________
	
	return( result*dim_inverse );				//correct for scaling factor in FFTs
}



__global__ void _gpu_extract_real_parts(float *dev_kernel_array, float *dev_temp, int rank0, int size1){

  int e = ((blockIdx.x * blockDim.x) + threadIdx.x);

	dev_kernel_array[rank0*size1 + e] = dev_temp[2*e];

	return;
}



void initialize_Gauss_quadrature_on_gpu(float *dev_qd_W_10, float *dev_qd_P_10, float *FD_cell_size){

	// initilize standard order 10 Gauss quadrature points and weights ______________________________
		float *std_qd_P_10 = (float*) calloc(10, sizeof(float));
		std_qd_P_10[0] = -0.97390652851717197f;
		std_qd_P_10[1] = -0.86506336668898498f;
		std_qd_P_10[2] = -0.67940956829902399f;
		std_qd_P_10[3] = -0.43339539412924699f;
		std_qd_P_10[4] = -0.14887433898163099f;
		std_qd_P_10[5] = -std_qd_P_10[4];
		std_qd_P_10[6] = -std_qd_P_10[3];
		std_qd_P_10[7] = -std_qd_P_10[2];
		std_qd_P_10[8] = -std_qd_P_10[1];
		std_qd_P_10[9] = -std_qd_P_10[0];
		float *host_qd_W_10 = (float*)calloc(10, sizeof(float));
		host_qd_W_10[0] = host_qd_W_10[9] = 0.066671344308687999f;
		host_qd_W_10[1] = host_qd_W_10[8] = 0.149451349150581f;
		host_qd_W_10[2] = host_qd_W_10[7] = 0.21908636251598201f;
		host_qd_W_10[3] = host_qd_W_10[6] = 0.26926671930999602f;
		host_qd_W_10[4] = host_qd_W_10[5] = 0.29552422471475298f;
	// ______________________________________________________________________________________________


	// Map the standard Gauss quadrature points to the used integration boundaries __________________
		float *host_qd_P_10 =  (float *) calloc (3*10, sizeof(float));
		get_Quad_Points(&host_qd_P_10[X*10], std_qd_P_10, 10, -0.5f*FD_cell_size[X], 0.5f*FD_cell_size[X]);
		get_Quad_Points(&host_qd_P_10[Y*10], std_qd_P_10, 10, -0.5f*FD_cell_size[Y], 0.5f*FD_cell_size[Y]);
		get_Quad_Points(&host_qd_P_10[Z*10], std_qd_P_10, 10, -0.5f*FD_cell_size[Z], 0.5f*FD_cell_size[Z]);
	// ______________________________________________________________________________________________

	// copy to the quadrature points and weights to the device ______________________________________
		memcpy_to_gpu (host_qd_W_10, dev_qd_W_10, 10);
		memcpy_to_gpu (host_qd_P_10, dev_qd_P_10, 3*10);
	// ______________________________________________________________________________________________

	free (std_qd_P_10);
	free (host_qd_P_10);
	free (host_qd_W_10);

	return;
}

void get_Quad_Points(float *gaussQP, float *stdGaussQP, int qOrder, double a, double b){

	int i;
	double A = (b-a)/2.0f; // coefficients for transformation x'= Ax+B
	double B = (a+b)/2.0f; // where x' is the new integration parameter

	gaussQP = (float *) calloc(qOrder, sizeof(float));

	for(i = 0; i < qOrder; i++)
		gaussQP[i] = A*stdGaussQP[i]+B;

	return;
}


#ifdef __cplusplus
}
#endif



// remove the following if code contains no errors for sure.

/*	
	float *host_temp = (float *)calloc(kernelStorageN, sizeof(float));			// temp array on host for storage of each component in real + i*complex format in serie (only for debugging purposes)
	float *host_temp2 = (float *)calloc(kernelStorageN/2, sizeof(float));	// temp array on host for storage of only the real components

	int testco1 = 0;
	int testco2 = 0;
	int testrang = 0;
	for (int i=0; i<testco1; i++)
		for (int j=i; j<testco2; j++)
			testrang ++;
	fprintf(stderr, "test co: %d, %d, testrang: %d\n\n", testco1, testco2, testrang);

	gpu_zero(dev_temp, kernelStorageN);
	hipDeviceSynchronize();
	_gpu_init_Greens_kernel_elements<<<gridsize, blocksize>>>(dev_temp, Nkernel[X], Nkernel[Y], Nkernel[Z], testco1, testco2, FD_cell_size[X], FD_cell_size[Y], FD_cell_size[Z], cst, repetition[X], repetition[Y], repetition[Z], dev_qd_P_10, dev_qd_W_10);
	hipDeviceSynchronize();

  memcpy_from_gpu(dev_temp, host_temp, kernelStorageN);
	hipDeviceSynchronize();
	fprintf(stderr, "\nkernel elements (untransformed), co: %d, %d:\n", testco1, testco2);
	for (int i=0; i<Nkernel[X]; i++){
		for (int j=0; j<Nkernel[Y]; j++){
			for (int k=0; k<gpu_pad_to_stride(Nkernel[Z]+2); k++){
				fprintf(stderr, "%e ", host_temp[i*Nkernel[Y]*gpu_pad_to_stride(Nkernel[Z]+2) + j*gpu_pad_to_stride(Nkernel[Z]+2) + k]);
			}
			fprintf(stderr, "\n");
		}
		fprintf(stderr, "\n");
	}
	
	gpu_plan3d_real_input_forward(kernel_plan, dev_temp);
	hipDeviceSynchronize();
	
  memcpy_from_gpu(dev_temp, host_temp, kernelStorageN);
	hipDeviceSynchronize();
	fprintf(stderr, "\nkernel elements (transformed), co: %d, %d:\n", testco1, testco2);
	for (int i=0; i<Nkernel[X]; i++){
		for (int j=0; j<Nkernel[Y]; j++){
			for (int k=0; k<gpu_pad_to_stride(Nkernel[Z]+2); k++){
				fprintf(stderr, "%e ", host_temp[i*Nkernel[Y]*gpu_pad_to_stride(Nkernel[Z]+2) + j*gpu_pad_to_stride(Nkernel[Z]+2) + k]);
			}
			fprintf(stderr, "\n");
		}
		fprintf(stderr, "\n");
	}

	_gpu_extract_real_parts<<<gridsize2, blocksize2>>>(&dev_kernel->list[testrang*kernelStorageN/2], dev_temp, 0, kernelStorageN/2);
	hipDeviceSynchronize();
	fprintf(stderr, "\nkernel elements (transformed, real parts), co: %d, %d:\n", testco1, testco2);
  memcpy_from_gpu(&dev_kernel->list[testrang*kernelStorageN/2], host_temp2, kernelStorageN/2);
	hipDeviceSynchronize();
		for (int i=0; i<Nkernel[X]; i++){
		for (int j=0; j<Nkernel[Y]; j++){
			for (int k=0; k<gpu_pad_to_stride(Nkernel[Z]+2)/2; k++){
				fprintf(stderr, "%e ", host_temp2[i*Nkernel[Y]*gpu_pad_to_stride(Nkernel[Z]+2)/2 + j*gpu_pad_to_stride(Nkernel[Z]+2)/2 + k]);
			}
			fprintf(stderr, "\n");
		}
		fprintf(stderr, "\n");
	}*/

