#include "hip/hip_runtime.h"
#include "gpufft2.h"
#include "gpuconv2.h"
#include "gputil.h"
#include "timer.h"
#include <stdio.h>
#include <assert.h>

#ifdef __cplusplus
extern "C" {
#endif

//_____________________________________________________________________________________________ copy/pad

/// @internal Does padding and unpadding, not necessarily by a factor 2
__global__ void _gpuconv2_copy_pad(float* source, float* dest, 
                                   int S1, int S2,                  ///< source sizes Y and Z
                                   int D1, int D2                   ///< destination size Y and Z
                                   ){
  int i = blockIdx.x;
  int j = blockIdx.y;
  int k = threadIdx.x;

  dest[(i*D1 + j)*D2 + k] = source[(i*S1 + j)*S2 + k];
}


void gpu_copy_pad(tensor* source, tensor* dest){
  
  assert(source->rank == 3);
  assert(  dest->rank == 3);
  
  // source must not be larger than dest
  for(int i=0; i<3; i++){
    assert(source->size[i] <= dest->size[i]);
  }
  
  int S0 = source->size[X];
  int S1 = source->size[Y];
  int S2 = source->size[Z];

  dim3 gridSize(S0, S1, 1); ///@todo generalize!
  dim3 blockSize(S2, 1, 1);
  gpu_checkconf(gridSize, blockSize);

  _gpuconv2_copy_pad<<<gridSize, blockSize>>>(source->list, dest->list, S1, S2, dest->size[1], dest->size[2]);
  hipDeviceSynchronize();
}

void gpu_copy_unpad(tensor* source, tensor* dest){
  
  assert(source->rank == 3);
  assert(  dest->rank == 3);
  
  // dest must not be larger than source
  for(int i=0; i<3; i++){
    assert(source->size[i] >= dest->size[i]);
  }
  
  int D0 = dest->size[X];
  int D1 = dest->size[Y];
  int D2 = dest->size[Z];

  dim3 gridSize(D0, D1, 1); ///@todo generalize!
  dim3 blockSize(D2, 1, 1);
  gpu_checkconf(gridSize, blockSize);

  _gpuconv2_copy_pad<<<gridSize, blockSize>>>(source->list, dest->list, source->size[1], source->size[2], D1, D2);
  hipDeviceSynchronize();
}

//_____________________________________________________________________________________________ kernel multiplication

/**
 * @internal 
 * Kernel is in interleaved complex format (imaginary part is zero and not read, but still stored),
 * and assummed symmetric.
 * The multiplication is in-place, fftMi is overwritten by fftHi
 */
__global__ void _gpu_kernel_mul_complex_inplace_symm(float* fftMx,  float* fftMy,  float* fftMz, 
                                                     float* fftKxx, float* fftKyy, float* fftKzz,
                                                     float* fftKyz, float* fftKxz, float* fftKxy){
  
  int e = 2 * ((blockIdx.x * blockDim.x) + threadIdx.x);
  
  // we some shared memory here, which saves an "8N" buffer in the global memory
  ///@todo coalescale read/writes
  float reMx = fftMx[e  ];
  float imMx = fftMx[e+1];
  
  float reMy = fftMy[e  ];
  float imMy = fftMy[e+1];
  
  float reMz = fftMz[e  ];
  float imMz = fftMz[e+1];
 
  float Kxx = fftKxx[e];
  float Kyy = fftKyy[e];
  float Kzz = fftKzz[e];
  
  float Kyz = fftKyz[e];
  float Kxz = fftKxz[e];
  float Kxy = fftKxy[e];
  
  fftMx[e  ] = reMx * Kxx + reMy * Kxy + reMz * Kxz;
  fftMx[e+1] = imMx * Kxx + imMy * Kxy + imMz * Kxz;
  
  fftMy[e  ] = reMx * Kxy + reMy * Kyy + reMz * Kyz;
  fftMy[e+1] = imMx * Kxy + imMy * Kyy + imMz * Kyz;
 
  fftMz[e  ] = reMx * Kxz + reMy * Kyz + reMz * Kzz;
  fftMz[e+1] = imMx * Kxz + imMy * Kyz + imMz * Kzz;

}


void gpu_kernel_mul_complex_inplace_symm(float* fftMx,  float* fftMy,  float* fftMz, 
                                         float* fftKxx, float* fftKyy, float* fftKzz,
                                         float* fftKyz, float* fftKxz, float* fftKxy,
                                         int nRealNumbers){
  
  timer_start("kernel_mul");
  assert(nRealNumbers > 0);
  assert(nRealNumbers % 2 == 0);
  
/*  int threadsPerBlock = 512;
  int blocks = (nRealNumbers/2) / threadsPerBlock;
  gpu_checkconf_int(blocks, threadsPerBlock);*/
  
   int gridSize = -1;
   int blockSize = -1;
   make1dconf(nRealNumbers/2, &gridSize, &blockSize);
   printf("gridsize = %d, blockSize = %d\n", gridSize, blockSize);
   
  _gpu_kernel_mul_complex_inplace_symm<<<gridSize, blockSize>>>(
                                      fftMx,  fftMy,  fftMz, 
                                      fftKxx, fftKyy, fftKzz,
                                      fftKyz, fftKxz, fftKxy);
  hipDeviceSynchronize();
  timer_stop("kernel_mul");
}

//_____________________________________________________________________________________________ convolution

void gpuconv2_exec(gpuconv2* conv, tensor* m, tensor* h){
  
  assertDevice(m->list);
  assertDevice(h->list);

  assert(m->rank == 4);
  assert(h->rank == 4);
  for(int i=0; i<4; i++){
    assert(m->size[i] == conv->m->size[i]);
    assert(h->size[i] == conv->h->size[i]);
  }
  
  ///@todo move to setMH()
  conv->m->list = m->list;                              // m, h, mComp and hComp are recycled tensors. We have to set their data each time.
  conv->h->list = h->list;                              // It would be cleaner to have them here as local variables, but this would
  for(int i=0; i<3; i++){                               // mean re-allocating them each time.
    conv->mComp[i]->list = &(m->list[conv->mComp[i]->len * i]);
    conv->hComp[i]->list = &(h->list[conv->hComp[i]->len * i]);
  }
 
  tensor** mComp = conv->mComp;                         // shorthand notations
  tensor** hComp = conv->hComp;
  tensor* fft1 = conv->fft1;
  tensor** fft1Comp = conv->fft1Comp;
  tensor** fft2Comp = conv->fft2Comp;
  
  //_____________________________________________________________________________________________ actual convolution
  
  gpu_zero_tensor(fft1);              // fft1 will now store the zero-padded magnetization
  gpu_zero_tensor(h);
  
  for(int i=0; i<3; i++){
    gpu_copy_pad(mComp[i], fft1Comp[i]);
  }
  
  hipDeviceSynchronize();  ///@todo many redundant syncs
  
  printf("conv2: Been here1\n");
  for(int i=0; i<3; i++){
    gpuFFT3dPlan_forward(conv->fftplan, fft1Comp[i], fft1Comp[i]);  ///@todo out-of-place
  }

  hipDeviceSynchronize();
  
  printf("conv2: Been here2\n");
  gpu_kernel_mul_complex_inplace_symm(fft1Comp[X]->list, fft1Comp[Y]->list, fft1Comp[Z]->list,
                                      conv->fftKernel[X][X]->list, conv->fftKernel[Y][Y]->list, conv->fftKernel[Z][Z]->list, 
                                      conv->fftKernel[Y][Z]->list, conv->fftKernel[X][Z]->list, conv->fftKernel[X][Y]->list,
                                      fft1Comp[X]->len);

  hipDeviceSynchronize();
  
  printf("conv2: Been here3\n");
  for(int i=0; i<3; i++){
    gpuFFT3dPlan_inverse(conv->fftplan, fft1Comp[i], fft1Comp[i]);  ///@todo out-of-place
  }
  
  hipDeviceSynchronize();
  
  for(int i=0; i<3; i++){
    gpu_copy_unpad(fft1Comp[i], hComp[i]);
  }
  
  hipDeviceSynchronize();
}



//_____________________________________________________________________________________________ load kernel

/**
 * the kernel gets normalized
 */
void gpuconv2_loadkernel5DSymm(gpuconv2* conv, tensor* kernel5D){
  
  int* paddedSize = conv->paddedSize;
  
  assert(kernel5D->rank == 5);
  assert(kernel5D->size[0] == 3);
  assert(kernel5D->size[1] == 3);
  assert(kernel5D->size[2+X] == paddedSize[X]);
  assert(kernel5D->size[2+Y] == paddedSize[Y]);
  assert(kernel5D->size[2+Z] == paddedSize[Z]);

  tensor* fftbuffer = conv->fft1Comp[0];
  tensor* KsdDev = new_gputensor(3, paddedSize); /// temp buffer, @todo: free! @todo: not a nice solution
  int* paddedStorageSize = fftbuffer->size;
  gpuFFT3dPlan* plan = new_gpuFFT3dPlan_padded(paddedSize, paddedSize); ///@todo change to paddedstoragesize when gpufft2 is updated

  float N = gpuFFT3dPlan_normalization(plan);
  
  // when the kernel is symmetric, this implementation does a bit redundant work
  for(int s=0; s<3; s++){
    for(int d=0; d<3; d++){
      tensor* Ksd = tensor_component(tensor_component(kernel5D, s), d);

      for(int i=0; i<Ksd->len; i++)
        Ksd->list[i] /= N;
      tensor_copy_to_gpu(Ksd, KsdDev);
      
      gpu_zero_tensor(conv->fftKernel[s][d]);
      gpu_copy_pad(KsdDev, conv->fftKernel[s][d]);
      gpuFFT3dPlan_forward(plan, conv->fftKernel[s][d], conv->fftKernel[s][d]);
    }
  }
  
}

// void gpuconv2_loadkernel(gpuconv2* conv, tensor* kernel){
//   fprintf(stderr, "loadkernel %d x %d x %d\n", kernel->size[2], kernel->size[3], kernel->size[4]);
//   
//   gpuconv2_checksize_kernel(conv, kernel);
//   gpu_plan3d_real_input* plan = new_gpu_plan3d_real_input(kernel->size[2], kernel->size[3], kernel->size[4], NO_ZERO_PAD);
//   float norm = 1.0/float(conv->fftplan->paddedN);
//   float* complex_kernel_ij = new_ram_array(conv->len_ft_kernel_ij);
//   for(int i=0; i<3; i++){
//       for(int j=0; j<3; j++){
// 	
// 	/// @todo !!!!!!!!!!!!!!!!!!!!!!! 
// 	//memcpy_r2c(tensor_get(kernel, 5, i, j, 0, 0, 0), complex_kernel_ij, conv->len_kernel_ij);
// 	
// 	//normalize
// 	for(int e=0; e<conv->len_ft_kernel_ij; e++){
// 	  complex_kernel_ij[e] *= norm;
// 	}
// 	memcpy_to_gpu(complex_kernel_ij, conv->ft_kernel[i][j], conv->len_ft_kernel_ij);
// 	//extract("kernel_ij", conv->ft_kernel[i][j], conv->paddedComplexSize);
// 	gpu_plan3d_real_input_forward(plan, conv->ft_kernel[i][j]);
// 	//extract("ft_kernel_ij", conv->ft_kernel[i][j], conv->paddedComplexSize);
//     }
//   }
//   free(complex_kernel_ij);
//   //delete_gpu_plan3d_real_input(plan);
// }




//_____________________________________________________________________________________________ new gpuconv2

gpuconv2* new_gpuconv2(int* size, int* kernelSize){
//   for(int i=0; i<3; i++){
//     assert(2*size[i] == kernelSize[i]); // generalize later
//   }
  
  gpuconv2* conv = (gpuconv2*)malloc(sizeof(gpuconv2));
  
  int* size4D = new int[4];
  size4D[0] = 3;
  size4D[1] = size[X];
  size4D[2] = size[Y];
  size4D[3] = size[Z];
  
  conv->paddedSize = kernelSize; ///@todo copy, to be sure (goes for all sizes)
  int* paddedSize = conv->paddedSize;
  
  int* paddedStorageSize = new int[3];  ///@todo obtain from fftplan instead
  paddedStorageSize[X] = paddedSize[X];
  paddedStorageSize[Y] = paddedSize[Y];
  paddedStorageSize[Z] = gpu_pad_to_stride(paddedSize[Z] + 2);
  
  int* paddedStorageSize4D = new int[4];
  paddedStorageSize4D[0] = 3;
  paddedStorageSize4D[1] = paddedStorageSize[X];
  paddedStorageSize4D[2] = paddedStorageSize[Y];
  paddedStorageSize4D[3] = paddedStorageSize[Z];
  

  // initialize the FFT plan
  ///@todo generalize !!
  int* zeroPad = new int[3];
  for(int i=0; i<3; i++){
    zeroPad[i] = 1; // todo !!
  }
  conv->fftplan = new_gpuFFT3dPlan_padded(size, kernelSize);	// it's important to FIRST initialize the fft plan because it stores the sizes used by other functions.
  
  conv->m = as_tensorN(NULL, 4, size4D);  // m->list will be set to whatever data is convolved at a certain time.
  conv->h = as_tensorN(NULL, 4, size4D);  // h->list will be set to whatever convolution destination used at a certain time.
  
  conv->fft1 = new_gputensor(4, paddedStorageSize4D);
  conv->fft2 = conv->fft1;  // in-place by default
  
  for(int i=0; i<3; i++){
    conv->mComp[i] = as_tensorN(NULL, 3, size); // note: as_tensor instead of as_tensorN did not gave compilation error and was very difficult to debug...
    conv->hComp[i] = as_tensorN(NULL, 3, size);
    
    conv->fft1Comp[i] = tensor_component(conv->fft1, i);
    conv->fft2Comp[i] = conv->fft1Comp[i]; // in-place by default
  }
  
   for(int i=0; i<3; i++)
    assert(conv->fftplan->paddedStorageSize[i] == conv->fft1Comp[0]->size[i]);

  // By default, the kernel is assumed to by symmetric. Should this not be the case, then the sub-diagonal elements should be separately allocated.
  conv->fftKernel[X][X] = new_gputensor(3, paddedStorageSize);
  conv->fftKernel[Y][Y] = new_gputensor(3, paddedStorageSize);
  conv->fftKernel[Z][Z] = new_gputensor(3, paddedStorageSize);
  
  conv->fftKernel[Y][Z] = new_gputensor(3, paddedStorageSize);
  conv->fftKernel[X][Z] = new_gputensor(3, paddedStorageSize);
  conv->fftKernel[X][Y] = new_gputensor(3, paddedStorageSize);
  
  ///@todo not storing the sub-diagonal elements does not work: they are somehow not loaded.
  conv->fftKernel[Z][Y] = new_gputensor(3, paddedStorageSize);//conv->fftKernel[Y][Z];
  conv->fftKernel[Z][X] = new_gputensor(3, paddedStorageSize);//conv->fftKernel[X][Z];
  conv->fftKernel[Y][X] = new_gputensor(3, paddedStorageSize);//conv->fftKernel[X][Y];

  ///@todo free some sizes
  return conv;
}



#ifdef __cplusplus
}
#endif