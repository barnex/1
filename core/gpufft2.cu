#include "gpufft2.h"
#include "gpufft.h"
#include "gputil.h"
#include "timer.h"
#include <stdio.h>
#include <assert.h>

#ifdef __cplusplus
extern "C" {
#endif


/**
 * Creates a new FFT plan for transforming the magnetization. 
 * Zero-padding in each dimension is optional, and rows with
 * only zero's are not transformed.
 * @todo on compute capability < 2.0, the first step is done serially...
 * @todo rename kernelsize -> paddedsize
 */
gpuFFT3dPlan* new_gpuFFT3dPlan_padded(int* size, int* paddedSize){
  
  int N0 = size[X];
  int N1 = size[Y];
  int N2 = size[Z];
  
  assert(paddedSize[X] > 0);
  assert(paddedSize[Y] > 1);
  assert(paddedSize[Z] > 1);
  
  gpuFFT3dPlan* plan = (gpuFFT3dPlan*)malloc(sizeof(gpuFFT3dPlan));
  
  plan->size = (int*)calloc(3, sizeof(int));    ///@todo not int* but int[3]
  plan->paddedSize = (int*)calloc(3, sizeof(int));
  plan->paddedStorageSize = (int*)calloc(3, sizeof(int));
  
//   int* paddedSize = plan->paddedSize;
  int* paddedStorageSize = plan->paddedStorageSize;
  
  plan->size[0] = N0; 
  plan->size[1] = N1; 
  plan->size[2] = N2;
  plan->N = N0 * N1 * N2;
  
  plan->paddedSize[X] = paddedSize[X];
  plan->paddedSize[Y] = paddedSize[Y];
  plan->paddedSize[Z] = paddedSize[Z];
  plan->paddedN = plan->paddedSize[0] * plan->paddedSize[1] * plan->paddedSize[2];
  
  plan->paddedStorageSize[X] = plan->paddedSize[X];
  plan->paddedStorageSize[Y] = plan->paddedSize[Y];
  plan->paddedStorageSize[Z] = gpu_pad_to_stride( plan->paddedSize[Z] +  2 );
  plan->paddedStorageN = paddedStorageSize[X] * paddedStorageSize[Y] * paddedStorageSize[Z];
  
  ///@todo check these sizes !
  gpu_safe( hipfftPlan1d(&(plan->fwPlanZ), plan->paddedSize[Z], HIPFFT_R2C, 1) );
  gpu_safe( hipfftPlan1d(&(plan->planY), plan->paddedSize[Y], HIPFFT_C2C, paddedStorageSize[Z] * size[X] / 2) );          // IMPORTANT: the /2 is necessary because the complex transforms have only half the amount of elements (the elements are now complex numbers)
  gpu_safe( hipfftPlan1d(&(plan->planX), plan->paddedSize[X], HIPFFT_C2C, paddedStorageSize[Z] * paddedSize[Y] / 2) );
  gpu_safe( hipfftPlan1d(&(plan->invPlanZ), plan->paddedSize[Z], HIPFFT_C2R, 1) );
  
  plan->transp = new_gpu_array(plan->paddedStorageN);
  
  return plan;
}


gpuFFT3dPlan* new_gpuFFT3dPlan(int* size){
  return new_gpuFFT3dPlan_padded(size, size); // when size == paddedsize, there is no padding
}


void gpuFFT3dPlan_forward(gpuFFT3dPlan* plan, tensor* input, tensor* output){
  timer_start("gpu_plan3d_real_input_forward_exec");
  
  assertDevice(input->list);
  assertDevice(output->list);

  assert(input == output); ///@todo works only in-place for now
  assert(input->rank == 3);
  assert(output->rank == 3);
  for(int i=0; i<3; i++){
    assert( input->size[i] == plan->paddedStorageSize[i]);
    assert(output->size[i] == plan->paddedStorageSize[i]);
  }
  
  int* size = plan->size;
  int* pSSize = plan->paddedStorageSize;
  int N0 = pSSize[X];
  int N1 = pSSize[Y];
  int N2 = pSSize[Z]/2; // we treat the complex data as an N0 x N1 x N2 x 2 array
  int N3 = 2;
  
  float* data = input->list;
  float* data2 = plan->transp; // both the transpose and FFT are out-of-place between data and data2
  
  printf("gpufft2, fw1\n");
	for(int i=0; i<size[X]; i++){
    for(int j=0; j<size[Y]; j++){
      float* rowIn  = &( input->list[i * pSSize[Y] * pSSize[Z] + j * pSSize[Z]]);
      float* rowOut = &(output->list[i * pSSize[Y] * pSSize[Z] + j * pSSize[Z]]);
      gpu_safe( hipfftExecR2C(plan->fwPlanZ, (hipfftReal*)rowIn,  (hipfftComplex*)rowOut) );
    }
  }
  hipDeviceSynchronize();
  
	printf("gpufft2, fw2\n");
  gpu_transposeYZ_complex(data, data2, N0, N1, N2*N3);                  // it's now in data2
  
	printf("gpufft2, fw3\n");
  gpu_safe( hipfftExecC2C(plan->planY, (hipfftComplex*)data2,  (hipfftComplex*)data2, HIPFFT_FORWARD) ); 
  hipDeviceSynchronize();

  // support for 2D transforms: do not transform if first dimension has size 1
  if(N0 > 1){
    gpu_transposeXZ_complex(data2, data, N0, N2, N1*N3); // size has changed due to previous transpose! // it's now in data2
    gpu_safe( hipfftExecC2C(plan->planX, (hipfftComplex*)data,  (hipfftComplex*)output->list, HIPFFT_FORWARD) ); // it's now again in data
    hipDeviceSynchronize();
  }
  hipDeviceSynchronize();
  timer_stop("gpu_plan3d_real_input_forward_exec");
}



void gpuFFT3dPlan_inverse(gpuFFT3dPlan* plan, tensor* input, tensor* output){
  timer_start("gpu_plan3d_real_input_inverse_exec");
  
  assertDevice(input->list);
  assertDevice(output->list);

  assert(input == output); ///@todo works only in-place for now
  assert(input->rank == 3);
  assert(output->rank == 3);
  for(int i=0; i<3; i++){
    assert( input->size[i] == plan->paddedStorageSize[i]);
    assert(output->size[i] == plan->paddedStorageSize[i]);
  }
  
  int* size = plan->size;
  int* pSSize = plan->paddedStorageSize;
  int N0 = pSSize[X];
  int N1 = pSSize[Y];
  int N2 = pSSize[Z]/2; // we treat the complex data as an N0 x N1 x N2 x 2 array
  int N3 = 2;
  
  float* data = input->list;
  float* data2 = plan->transp; // both the transpose and FFT are out-of-place between data and data2

	printf("gpufft2, inv3\n");
  if (N0 > 1){
    // input data is XZ transposed and stored in data, FFTs on X-arrays out of place towards data2
    gpu_safe( hipfftExecC2C(plan->planX, (hipfftComplex*)data,  (hipfftComplex*)data2, HIPFFT_BACKWARD) ); // it's now in data2
    hipDeviceSynchronize();
    gpu_transposeXZ_complex(data2, data, N1, N2, N0*N3); // size has changed due to previous transpose! // it's now in data
  }

	printf("gpufft2, inv2\n");
	gpu_safe( hipfftExecC2C(plan->planY, (hipfftComplex*)data,  (hipfftComplex*)data2, HIPFFT_BACKWARD) ); // it's now again in data2
  hipDeviceSynchronize();

  gpu_transposeYZ_complex(data2, data, N0, N2, N1*N3);                 

	printf("gpufft2, inv1\n");
	for(int i=0; i<size[X]; i++){
    for(int j=0; j<size[Y]; j++){
      float* rowIn  = &( input->list[i * pSSize[Y] * pSSize[Z] + j * pSSize[Z]]);
      float* rowOut = &(output->list[i * pSSize[Y] * pSSize[Z] + j * pSSize[Z]]);
      gpu_safe( hipfftExecC2R(plan->invPlanZ, (hipfftComplex*)rowIn, (hipfftReal*)rowOut) ); 
    }
  }
  hipDeviceSynchronize();
  
  timer_stop("gpu_plan3d_real_input_inverse_exec");
}


int gpuFFT3dPlan_normalization(gpuFFT3dPlan* plan){
  return plan->paddedSize[X] * plan->paddedSize[Y] * plan->paddedSize[Z];
}

//_____________________________________________________________________________________________ transpose

void gpu_tensor_transposeYZ_complex(tensor* source, tensor* dest){
	
  assert(source != dest);                       // must be out-of-place
  assert(source->rank == 3);
  assert(dest->rank == 3);
  assert(dest->size[Y] == source->size[Z]/2);   // interleaved complex format
  assert(dest->size[Z] == source->size[Y]*2);
  
  timer_start("transposeYZ");
  
  // we treat the complex array as a N0 x N1 x N2 x 2 real array
  // after transposing it becomes N0 x N2 x N1 x 2
  int N0 = source->size[X];
  int N1 = source->size[Y];
  int N2 = source->size[Z] / 2;
  //int N3 = 2; // not used
  
  dim3 gridsize(N0, N1, 1);  ///@todo generalize!
  dim3 blocksize(N2, 1, 1);
  gpu_checkconf(gridsize, blocksize);
  _gpu_transposeYZ_complex<<<gridsize, blocksize>>>(source->list, dest->list, N0, N1, N2);
  hipDeviceSynchronize();
  
  timer_stop("transposeYZ");
}

void gpu_tensor_transposeXZ_complex(tensor* source, tensor* dest){
  assert(source != dest);                       // must be out-of-place
  assert(source->rank == 3);
  assert(dest->rank == 3);
  assert(dest->size[X] == source->size[Z]/2);   // interleaved complex format
  assert(dest->size[Z] == source->size[X]*2);
  
  timer_start("transposeXZ");
  
  // we treat the complex array as a N0 x N1 x N2 x 2 real array
  // after transposing it becomes N0 x N2 x N1 x 2
  int N0 = source->size[X];
  int N1 = source->size[Y];
  int N2 = source->size[Z] / 2;
  //int N3 = 2; // not used
  
  dim3 gridsize(N0, N1, 1);  ///@todo generalize!
  dim3 blocksize(N2, 1, 1);
  gpu_checkconf(gridsize, blocksize);
  _gpu_transposeXZ_complex<<<gridsize, blocksize>>>(source->list, dest->list, N0, N1, N2);
  hipDeviceSynchronize();
  
  timer_stop("transposeXZ");
}


// //_____________________________________________________________________________________________ exec plan
// 
// void gpu_plan3d_real_input_forward(gpu_plan3d_real_input* plan, float* data){
//   timer_start("gpu_plan3d_real_input_forward_exec");
// 
//   int* size = plan->size;
//   int* pSSize = plan->paddedStorageSize;
//   int N0 = pSSize[X];
//   int N1 = pSSize[Y];
//   int N2 = pSSize[Z]/2; // we treat the complex data as an N0 x N1 x N2 x 2 array
//   int N3 = 2;
//   
//   float* data2 = plan->transp; // both the transpose and FFT are out-of-place between data and data2
//   
//   for(int i=0; i<size[X]; i++){
//     for(int j=0; j<size[Y]; j++){
//       float* row = &(data[i * pSSize[Y] * pSSize[Z] + j * pSSize[Z]]);
//       gpu_safe( hipfftExecR2C(plan->fwPlanZ, (hipfftReal*)row,  (hipfftComplex*)row) ); // all stays in data
//     }
//   }
//   hipDeviceSynchronize();
//   
//   gpu_transposeYZ_complex(data, data2, N0, N1, N2*N3);					// it's now in data2
//   gpu_safe( hipfftExecC2C(plan->planY, (hipfftComplex*)data2,  (hipfftComplex*)data2, HIPFFT_FORWARD) ); // it's now again in data
//   hipDeviceSynchronize();
//   
//   gpu_transposeXZ_complex(data2, data, N0, N2, N1*N3); // size has changed due to previous transpose! // it's now in data2
//   gpu_safe( hipfftExecC2C(plan->planX, (hipfftComplex*)data,  (hipfftComplex*)data, HIPFFT_FORWARD) ); // it's now again in data
//   hipDeviceSynchronize();
//   
//   timer_stop("gpu_plan3d_real_input_forward_exec");
// }
// 
// void gpu_plan3d_real_input_inverse(gpu_plan3d_real_input* plan, float* data){
//   
// }
// 
// void delete_gpu_plan3d_real_input(gpu_plan3d_real_input* plan){
//   
// 	gpu_safe( hipfftDestroy(plan->fwPlanZ) );
// 	gpu_safe( hipfftDestroy(plan->invPlanZ) );
// 	gpu_safe( hipfftDestroy(plan->planY) );
// 	gpu_safe( hipfftDestroy(plan->planX) );
// 
// 	gpu_safe( hipFree(plan->transp) ); 
// 	gpu_safe( hipFree(plan->size) );
// 	gpu_safe( hipFree(plan->paddedSize) );
// 	gpu_safe( hipFree(plan->paddedStorageSize) );
// 	free(plan);
// 
// }


#ifdef __cplusplus
}
#endif