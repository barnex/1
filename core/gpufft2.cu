#include "gpufft2.h"
#include "gpufft.h"
#include "gputil.h"
#include "timer.h"
#include <stdio.h>
#include <assert.h>

#ifdef __cplusplus
extern "C" {
#endif


/**
 * Creates a new FFT plan for transforming the magnetization. 
 * Zero-padding in each dimension is optional, and rows with
 * only zero's are not transformed.
 * @todo on compute capability < 2.0, the first step is done serially...
 */
gpuFFT3dPlan* new_gpuFFT3dPlan(int* size,       ///< size of real input data (3D)
                               int* kernelsize  ///< size of the kernel (3D). Should be at least the size of the input data. If the kernel is larger, the input data is assumed to be padded with zero's which are efficiently handled by the FFT
                               );

/**
 * Forward (real-to-complex) transform.
 */
void gpuFFT3dPlan_forward(gpuFFT3dPlan* plan,       ///< the plan to be executed
                          tensor* input,            ///< input data, it's size should match the strided "half complex" format (=plan->paddedStorageSize)
                          tensor* output            ///< output data, may be equal to input for in-place transforms.
                          );

/**
 * Backward (complex-to-real) transform.
 */
void gpuFFT3dPlan_inverse(gpuFFT3dPlan* plan,       ///< the plan to be executed
                          tensor* input,            ///< input data, may be equal to output for in-place transforms.
                          tensor* output            ///< output data, it's size should match the strided "half complex" format (=plan->paddedStorageSize)
                          );

/**
 * @internal
 * Swaps the Y and Z components of a 3D array of complex numbers.
 * N0 x N1 x N2/2 complex numbers are stored as N0 x N1 x N2 interleaved real numbers.
 */
void gpu_tensor_transposeYZ_complex(tensor* source, ///< source data, size N0 x N1 x (2*N2)
                                    tensor* dest   ///< destination data, size N0 x N2 x (2*N1)
                             );
/**
 * @internal
 * @see gpu_transposeYZ_complex()
 */
void gpu_tensor_transposeXZ_complex(tensor* source, ///< source data, size N0 x N1 x (2*N2)
                                    tensor* dest   ///< destination data, size N2 x N1 x (2*N0)
                             );

                             
                             

// gpu_plan3d_real_input* new_gpu_plan3d_real_input(int N0, int N1, int N2, int* zero_pad){
//   assert(N0 > 1);
//   assert(N1 > 1);
//   assert(N2 > 1);
//   
//   gpu_plan3d_real_input* plan = (gpu_plan3d_real_input*)malloc(sizeof(gpu_plan3d_real_input));
//   
//   plan->size = (int*)calloc(3, sizeof(int));
//   plan->paddedSize = (int*)calloc(3, sizeof(int));
//   plan->paddedStorageSize = (int*)calloc(3, sizeof(int));
//     
//   int* size = plan->size;
//   int* paddedSize = plan->paddedSize;
//   int* paddedStorageSize = plan->paddedStorageSize;
//   
//   plan->size[0] = N0; 
//   plan->size[1] = N1; 
//   plan->size[2] = N2;
//   plan->N = N0 * N1 * N2;
//   
//  
//   plan->paddedSize[X] = (1 + zero_pad[X]) * N0; 
//   plan->paddedSize[Y] = (1 + zero_pad[Y]) * N1; 
//   plan->paddedSize[Z] = (1 + zero_pad[Z]) * N2;
//   plan->paddedN = plan->paddedSize[0] * plan->paddedSize[1] * plan->paddedSize[2];
//   
//   plan->paddedStorageSize[X] = plan->paddedSize[X];
//   plan->paddedStorageSize[Y] = plan->paddedSize[Y];
// //  plan->paddedStorageSize[Z] = plan->paddedSize[Z] +  gpu_stride_float();   ///@todo aanpassen!!
//   plan->paddedStorageSize[Z] = gpu_pad_to_stride( plan->paddedSize[Z] +  2 );
//   plan->paddedStorageN = paddedStorageSize[X] * paddedStorageSize[Y] * paddedStorageSize[Z];
//   
//   gpu_safe( hipfftPlan1d(&(plan->fwPlanZ), plan->paddedSize[Z], HIPFFT_R2C, 1) );
//   gpu_safe( hipfftPlan1d(&(plan->planY), plan->paddedSize[Y], HIPFFT_C2C, paddedStorageSize[Z] * size[X]) );
//   gpu_safe( hipfftPlan1d(&(plan->planX), plan->paddedSize[X], HIPFFT_C2C, paddedStorageSize[Z] * paddedSize[Y]) );
//   gpu_safe( hipfftPlan1d(&(plan->invPlanZ), plan->paddedSize[Z], HIPFFT_C2R, 1) );
//   
//   plan->transp = new_gpu_array(plan->paddedStorageN);
//   
//   return plan;
// }
// 
// //_____________________________________________________________________________________________ transpose
// 



                             
// void gpu_tensor_transposeXZ_complex(tensor* source, ///< source data, size N0 x N1 x (2*N2)
//                                     tensor* dest   ///< destination data, size N2 x N1 x (2*N0)
//                              );

// 
// void gpu_transposeXZ_complex(float* source, float* dest, int N0, int N1, int N2){
//   timer_start("transposeXZ"); /// @todo section is double-timed with FFT exec
//   
//   assert(source != dest); // must be out-of-place
//   
//   // we treat the complex array as a N0 x N1 x N2 x 2 real array
//   // after transposing it becomes N0 x N2 x N1 x 2
//   N2 /= 2;
//   //int N3 = 2;
//   
//   dim3 gridsize(N0, N1, 1);	///@todo generalize!
//   dim3 blocksize(N2, 1, 1);
//   gpu_checkconf(gridsize, blocksize);
//   _gpu_transposeXZ_complex<<<gridsize, blocksize>>>(source, dest, N0, N1, N2);
//   hipDeviceSynchronize();
//   
//   timer_stop("transposeXZ");
// }
// 
// //_____________________________________________________________________________________________
// 

void gpu_tensor_transposeYZ_complex(tensor* source, tensor* dest){
  assert(source != dest);                       // must be out-of-place
  assert(source->rank == 3);
  assert(dest->rank == 3);
  assert(dest->size[Y] == source->size[Z]/2);   // interleaved complex format
  assert(dest->size[Z] == source->size[Y]*2);
  
  timer_start("transposeYZ");
  
  // we treat the complex array as a N0 x N1 x N2 x 2 real array
  // after transposing it becomes N0 x N2 x N1 x 2
  int N0 = source->size[X];
  int N1 = source->size[Y];
  int N2 = source->size[Z] / 2;
  int N3 = 2;
  
  dim3 gridsize(N0, N1, 1);  ///@todo generalize!
  dim3 blocksize(N2, 1, 1);
  gpu_checkconf(gridsize, blocksize);
  _gpu_transposeYZ_complex<<<gridsize, blocksize>>>(source->list, dest->list, N0, N1, N2);
  hipDeviceSynchronize();
  
  timer_stop("transposeYZ");
}

void gpu_tensor_transposeXZ_complex(tensor* source, tensor* dest){
  assert(source != dest);                       // must be out-of-place
  assert(source->rank == 3);
  assert(dest->rank == 3);
  assert(dest->size[X] == source->size[Z]/2);   // interleaved complex format
  assert(dest->size[Z] == source->size[X]*2);
  
  timer_start("transposeXZ");
  
  // we treat the complex array as a N0 x N1 x N2 x 2 real array
  // after transposing it becomes N0 x N2 x N1 x 2
  int N0 = source->size[X];
  int N1 = source->size[Y];
  int N2 = source->size[Z] / 2;
  int N3 = 2;
  
  dim3 gridsize(N0, N1, 1);  ///@todo generalize!
  dim3 blocksize(N2, 1, 1);
  gpu_checkconf(gridsize, blocksize);
  _gpu_transposeXZ_complex<<<gridsize, blocksize>>>(source->list, dest->list, N0, N1, N2);
  hipDeviceSynchronize();
  
  timer_stop("transposeXZ");
}

// 
// void gpu_transposeYZ_complex(float* source, float* dest, int N0, int N1, int N2){
//   timer_start("transposeYZ");
//   
//   assert(source != dest); // must be out-of-place
//   
//   // we treat the complex array as a N0 x N1 x N2 x 2 real array
//   // after transposing it becomes N0 x N2 x N1 x 2
//   N2 /= 2;
//   //int N3 = 2;
//   
//   dim3 gridsize(N0, N1, 1);	///@todo generalize!
//   dim3 blocksize(N2, 1, 1);
//   gpu_checkconf(gridsize, blocksize);
//   _gpu_transposeYZ_complex<<<gridsize, blocksize>>>(source, dest, N0, N1, N2);
//   hipDeviceSynchronize();
//   
//   timer_stop("transposeYZ");
// }
// 
// //_____________________________________________________________________________________________ exec plan
// 
// void gpu_plan3d_real_input_forward(gpu_plan3d_real_input* plan, float* data){
//   timer_start("gpu_plan3d_real_input_forward_exec");
// 
//   int* size = plan->size;
//   int* pSSize = plan->paddedStorageSize;
//   int N0 = pSSize[X];
//   int N1 = pSSize[Y];
//   int N2 = pSSize[Z]/2; // we treat the complex data as an N0 x N1 x N2 x 2 array
//   int N3 = 2;
//   
//   float* data2 = plan->transp; // both the transpose and FFT are out-of-place between data and data2
//   
//   for(int i=0; i<size[X]; i++){
//     for(int j=0; j<size[Y]; j++){
//       float* row = &(data[i * pSSize[Y] * pSSize[Z] + j * pSSize[Z]]);
//       gpu_safe( hipfftExecR2C(plan->fwPlanZ, (hipfftReal*)row,  (hipfftComplex*)row) ); // all stays in data
//     }
//   }
//   hipDeviceSynchronize();
//   
//   gpu_transposeYZ_complex(data, data2, N0, N1, N2*N3);					// it's now in data2
//   gpu_safe( hipfftExecC2C(plan->planY, (hipfftComplex*)data2,  (hipfftComplex*)data2, HIPFFT_FORWARD) ); // it's now again in data
//   hipDeviceSynchronize();
//   
//   gpu_transposeXZ_complex(data2, data, N0, N2, N1*N3); // size has changed due to previous transpose! // it's now in data2
//   gpu_safe( hipfftExecC2C(plan->planX, (hipfftComplex*)data,  (hipfftComplex*)data, HIPFFT_FORWARD) ); // it's now again in data
//   hipDeviceSynchronize();
//   
//   timer_stop("gpu_plan3d_real_input_forward_exec");
// }
// 
// void gpu_plan3d_real_input_inverse(gpu_plan3d_real_input* plan, float* data){
//   timer_start("gpu_plan3d_real_input_inverse_exec");
// 
//   int* size = plan->size;
//   int* pSSize = plan->paddedStorageSize;
//   int N0 = pSSize[X];
//   int N1 = pSSize[Y];
//   int N2 = pSSize[Z]/2; // we treat the complex data as an N0 x N1 x N2 x 2 array
//   int N3 = 2;
//   
//   float* data2 = plan->transp; // both the transpose and FFT are out-of-place between data and data2
// 
// 	// input data is XZ transpozed and stored in data, FFTs on X-arrays out of place towards data2
//   gpu_safe( hipfftExecC2C(plan->planX, (hipfftComplex*)data,  (hipfftComplex*)data2, HIPFFT_BACKWARD) ); // it's now in data2
//   hipDeviceSynchronize();
// //  gpu_transposeXZ_complex(data2, data, N0, N2, N1*N3); // size has changed due to previous transpose! // it's now in data
//   gpu_transposeXZ_complex(data2, data, N1, N2, N0*N3); // size has changed due to previous transpose! // it's now in data
//   
//   gpu_safe( hipfftExecC2C(plan->planY, (hipfftComplex*)data,  (hipfftComplex*)data2, HIPFFT_BACKWARD) ); // it's now again in data2
//   hipDeviceSynchronize();
// //  gpu_transposeYZ_complex(data2, data, N0, N1, N2*N3);					// it's now in data
//   gpu_transposeYZ_complex(data2, data, N0, N2, N1*N3);					// it's now in data
// 
//   for(int i=0; i<size[X]; i++){
//     for(int j=0; j<size[Y]; j++){
//       float* row = &(data[i * pSSize[Y] * pSSize[Z] + j * pSSize[Z]]);
//       gpu_safe( hipfftExecC2R(plan->invPlanZ, (hipfftComplex*)row, (hipfftReal*)row) ); // all stays in data
//     }
//   }
//   hipDeviceSynchronize();
//   
//   timer_stop("gpu_plan3d_real_input_inverse_exec");
// }
// 
// void delete_gpu_plan3d_real_input(gpu_plan3d_real_input* plan){
//   
// 	gpu_safe( hipfftDestroy(plan->fwPlanZ) );
// 	gpu_safe( hipfftDestroy(plan->invPlanZ) );
// 	gpu_safe( hipfftDestroy(plan->planY) );
// 	gpu_safe( hipfftDestroy(plan->planX) );
// 
// 	gpu_safe( hipFree(plan->transp) ); 
// 	gpu_safe( hipFree(plan->size) );
// 	gpu_safe( hipFree(plan->paddedSize) );
// 	gpu_safe( hipFree(plan->paddedStorageSize) );
// 	free(plan);
// 
// }


#ifdef __cplusplus
}
#endif