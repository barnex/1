#include "hip/hip_runtime.h"
#include "gpusim.h"
#include <stdio.h>
#include <assert.h>

#ifdef __cplusplus
extern "C" {
#endif

void print(const char* msg, tensor* t){
  printf("%s:\n", msg);
  format_tensor(t, stdout);
}

//_____________________________________________________________________________________________ sim

void gpusim_updateh(gpusim* sim){
  gpu_zero(sim->h, sim->len_h);								// zero-out field (h) components
  //for(int i=0; i<3; i++){								// transform and convolve per magnetization component m_i
    gpu_zero(sim->ft_m_i, sim->len_ft_m_i);						// zero-out the padded magnetization buffer first
    gpu_copy_pad_r2c(sim->m, sim->ft_m_i, sim->size[0], sim->size[1], sim->size[2]);	//copy mi into the padded magnetization buffer, converting to complex format
  //}
}

//_____________________________________________________________________________________________ load / store data

void gpusim_checksize_m(gpusim* sim, tensor* m){
   // m should be a rank 4 tensor with size 3 x N0 x N1 x N2
  assert(m->rank == 4);
  assert(m->size[0] == 3); 
  for(int i=0; i<3; i++){ assert(m->size[i+1] == sim->size[i]); }
}

void gpusim_loadm(gpusim* sim, tensor* m){
  gpusim_checksize_m(sim, m); 
  memcpy_to_gpu(m->list, sim->m, sim->len_m);
}

void gpusim_storem(gpusim* sim, tensor* m){
   gpusim_checksize_m(sim, m);
   memcpy_from_gpu(sim->m, m->list, sim->len_m);
}

//_____________________________________________________________________________________________ kernel

void gpusim_checksize_kernel(gpusim* sim, tensor* kernel){
  // kernel should be rank 5 tensor with size 3 x 3 x 2*N0 x 2xN1 x 2xN2 (could be reduced a bit)
  assert(kernel->rank == 5);
  assert(kernel->size[0] == 3);
  assert(kernel->size[1] == 3);
  for(int i=0; i<3; i++){ assert(kernel->size[i+2] == 2 * sim->size[i]); }
}

void gpusim_alloc_ft_kernel(gpusim* sim){
  sim->ft_kernel = (float***)calloc(3, sizeof(float**));
  for(int i=0; i<3; i++){ 
    sim->ft_kernel[i] = (float**)calloc(3, sizeof(float*));
    for(int j=0; j<3; j++){
      sim->ft_kernel[i][j] = new_gpu_array(sim->len_ft_kernel_ij);
    }
  }
}

void gpusim_loadkernel(gpusim* sim, tensor* kernel){
  gpusim_checksize_kernel(sim, kernel);
  gpusim_c2cplan* plan = new_gpusim_c2cplan(kernel->size[2], kernel->size[3], kernel->size[4]);
  float* complex_kernel_ij = new_ram_array(sim->len_ft_kernel_ij);
  for(int i=0; i<3; i++){
      for(int j=0; j<3; j++){
	memcpy_r2c(tensor_get(kernel, 5, i, j, 0, 0, 0), complex_kernel_ij, sim->len_kernel_ij);
	memcpy_to_gpu(complex_kernel_ij, sim->ft_kernel[i][j], sim->len_ft_kernel_ij);
	gpusim_c2cplan_exec(plan, sim->ft_kernel[i][j], HIPFFT_FORWARD);
    }
  }
  free(complex_kernel_ij);
  delete_gpusim_c2cplan(plan);
}

//_____________________________________________________________________________________________ new

gpusim* new_gpusim(int N0, int N1, int N2, tensor* kernel){
  gpusim* sim = (gpusim*)malloc(sizeof(gpusim));
  
  // init sizes
  sim->size = (int*)calloc(3, sizeof(int));
  sim->size[0] = N0; sim->size[1] = N1; sim->size[2] = N2;
  sim->N = N0 * N1 * N2;
  
  // init kernel
  sim->len_kernel_ij = kernel->size[2] * kernel->size[3] * kernel->size[4];		//the length of each kernel component K[i][j] (eg: Kxy)
  sim->len_ft_kernel_ij = 2 * sim->len_kernel_ij;					//the length of each FFT'ed kernel component ~K[i][j] (eg: ~Kxy)
  gpusim_alloc_ft_kernel(sim);
  gpusim_loadkernel(sim, kernel);
  
  // init magnetization arrays
  sim->len_m = 3 * sim->N;
  sim->m = new_gpu_array(sim->len_m);
  sim->len_m_comp = sim->N; 
  sim->m_comp = (float**)calloc(3, sizeof(float*));
  for(int i=0; i<3; i++){ 
    sim->m_comp[i] = &(sim->m[i * sim->len_m_comp]); 
  }
  sim->len_ft_m_i = sim->len_ft_kernel_ij;
  sim->ft_m_i = new_gpu_array(sim->len_ft_m_i);
  
  // init h
  sim->len_h = sim->len_m;
  sim->h = new_gpu_array(sim->len_h);
  sim->len_h_comp = sim->N; 
  sim->h_comp = (float**)calloc(3, sizeof(float*));
  for(int i=0; i<3; i++){ 
    sim->h_comp[i] = &(sim->h[i * sim->len_h_comp]); 
  }
  sim->len_ft_h_i = sim->len_ft_kernel_ij;
  sim->ft_h_i = new_gpu_array(sim->len_ft_h_i);
  
  return sim;
}

//_____________________________________________________________________________________________ FFT

gpusim_c2cplan* new_gpusim_c2cplan(int N0, int N1, int N2){
  gpusim_c2cplan* plan = (gpusim_c2cplan*) malloc(sizeof(gpusim_c2cplan));
  gpusim_safe( hipfftPlan3d(&(plan->handle), N0, N1, N2, HIPFFT_C2C) );
  return plan;
}

void gpusim_c2cplan_exec(gpusim_c2cplan* plan, float* data, int direction){
  gpusim_safe( 
    hipfftExecC2C(plan->handle, (hipfftComplex*)data, (hipfftComplex*)data, direction) 
  );
}

void delete_gpusim_c2cplan(gpusim_c2cplan* plan){
  //gpusim_safe( hipFree(plan->gpudata) );
  // TODO: free handle
  free(plan);
}

//_____________________________________________________________________________________________ data management

__global__ void _gpu_copy_pad_r2c(float* source, float* dest, int N0, int N1, int N2){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  
  dest[i*2*N1*2*N2 + j*2*N1 + 2*k] = source[i*N1*N2 + j*N1 + k];
  dest[i*2*N1*2*N2 + j*2*N1 + 2*k + 1] = 0.;
  
}

void gpu_copy_pad_r2c(float* source, float* dest, int N0, int N1, int N2){
  assert(N0 % 16 == 0);
  assert(N1 % 16 == 0);
  
  dim3 threadsPerBlock(16, 16, 1);
  dim3 numBlocks(N0 / threadsPerBlock.x, N1 / threadsPerBlock.y, N2 / threadsPerBlock.z);
  _gpu_copy_pad_r2c<<<numBlocks, threadsPerBlock>>>(source, dest, N0, N1, N2);

}

int gpu_len(int size){
  assert(size > 0);
  int gpulen = ((size-1)/threadsPerBlock + 1) * threadsPerBlock;
  assert(gpulen % threadsPerBlock == 0);
  assert(gpulen > 0);
  return gpulen;
}

__global__ void _gpu_zero(float* list){
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  list[i] = 0.;
}


void gpu_zero(float* data, int nElements){
  assert(nElements > 0);
  int blocks = nElements / threadsPerBlock;
  _gpu_zero<<<blocks, threadsPerBlock>>>(data);
}

void memcpy_to_gpu(float* source, float* dest, int nElements){
  assert(nElements > 0);
  int status = hipMemcpy(dest, source, nElements*sizeof(float), hipMemcpyHostToDevice);
  if(status != hipSuccess){
    fprintf(stderr, "CUDA could not copy %d floats from host addres %p to device addres %p\n", nElements, source, dest);
    gpusim_safe(status);
  }
}


void memcpy_from_gpu(float* source, float* dest, int nElements){
  assert(nElements > 0);
  int status = hipMemcpy(dest, source, nElements*sizeof(float), hipMemcpyDeviceToHost);
  if(status != hipSuccess){
    fprintf(stderr, "CUDA could not copy %d floats from device addres %p to host addres %p\n", nElements, source, dest);
    gpusim_safe(status);
  }
}

// does not seem to work.. 
void memcpy_gpu_to_gpu(float* source, float* dest, int nElements){
  assert(nElements > 0);
  int status = hipMemcpy(dest, source, nElements*sizeof(float), hipMemcpyHostToHost);
  if(status != hipSuccess){
    fprintf(stderr, "CUDA could not copy %d floats from host addres %p to host addres %p\n", nElements, source, dest);
    gpusim_safe(status);
  }
}

// todo: we need hipMalloc3D for better alignment!
float* new_gpu_array(int size){
  assert(size > 0);
  assert(size % threadsPerBlock == 0);
  float* array = NULL;
  int status = hipMalloc((void**)(&array), size * sizeof(float));
  if(status != hipSuccess){
    fprintf(stderr, "CUDA could not allocate %d floats\n", size);
    gpusim_safe(status);
  }
  //assert(array != NULL); // strange: it seems cuda can return 0 as a valid address?? 
  if(array == 0){
    fprintf(stderr, "hipMalloc(%p, %ld) returned null without error status, retrying...\n", (void**)(&array), size * sizeof(float));
    abort();
  }
  return array;
}

float* new_ram_array(int size){
  assert(size > 0);
  float* array = (float*)calloc(size, sizeof(float));
  if(array == NULL){
    fprintf(stderr, "could not allocate %d floats in main memory\n", size);
    abort();
  }
  return array;
}

void memcpy_r2c(float* source, float* dest, int nReal){
  for(int i=0; i<nReal; i++){
    dest[2*i]     = source[i];
    dest[2*i + 1] = 0.;
  }
}

//_____________________________________________________________________________________________ misc

void gpusim_safe(int status){
  if(status != hipSuccess){
    fprintf(stderr, "received CUDA error: %s\n", hipGetErrorString((hipError_t)status));
    abort();
  }
}

#ifdef __cplusplus
}
#endif