/*
 * Example of 3D cuda FFT. Copies some random data to the GPU where it is transformed back and forth, and copied back to the main RAM. The FFT error is checked.
 */


#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <assert.h>

double sqr(double x){
  return x*x;
}

int main(){
 
 int N0 = 128, N1 = 128, N2 = 8;	// size of the data to be transformed
 int N = 2 * N0 * N1 * N2;		// size of complex array
 
 float* data_host, *data_dev;		// the original data, on the host (PC) and a copy on the device (GPU)
 float* transf_dev; 			// the transformed data, on the device
 float* transf2_host, *transf2_dev; 	// transformed+backtransformed data, on the device and a copy on host
 
 data_host = (float*)malloc(N * sizeof(float));		
 hipMalloc((void**)&data_dev, N * sizeof(float));	
 transf2_host = (float*)malloc((N) * sizeof(float));
 hipMalloc((void**)&transf_dev, (N) * sizeof(float));
 hipMalloc((void**)&transf2_dev, N * sizeof(float));
 
 int i;
 double rmserror = 0;
 
 hipfftHandle plan;			
 hipfftPlan3d(&plan, N0, N1, N2, HIPFFT_C2C);
    
 for(i=0; i<N; i++){				// make some data on the host
  data_host[i] = (rand() % 10000) / 10000.0;
 }
 
  hipMemcpy(data_dev, data_host, N * sizeof(float), hipMemcpyHostToDevice);	// copy data to the device
  
  hipfftExecC2C(plan, (hipfftComplex*)data_dev, (hipfftComplex*)transf_dev, HIPFFT_FORWARD);
  hipfftExecC2C(plan, (hipfftComplex*)transf_dev, (hipfftComplex*)transf2_dev, HIPFFT_BACKWARD);

  
  hipMemcpy(transf2_host, transf2_dev, N * sizeof(float), hipMemcpyDeviceToHost); // copy back to host
 
  // check RMS error of transform+backtransform
  for(i=0; i<N; i++){
    //printf("%f\n", transf2_host[i] / N);
    rmserror += sqr(data_host[i] - transf2_host[i] / (N0 * N1 * N2) );	
    //assert(transf2_host[i] != 0.0);
  }
  rmserror = sqrt(rmserror);
  printf("FFT error: %lf\n", rmserror);
  assert(rmserror < 1E-3);

  // clean up:
 hipfftDestroy(plan);
 hipFree(data_dev);
 hipFree(transf2_dev);
 hipFree(transf_dev);
 free(data_host);
 free(transf2_host);
 
  return 0;
}