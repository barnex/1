
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(){

  int i;

  float* ah;
  float* bh;
  float* ad;
  float* bd;

  int n = 10;
  int nbytes = n*sizeof(float);

  ah = (float*)malloc(nbytes);
  bh = (float*)malloc(nbytes);

  for(i=0; i<n; i++){
    ah[i] = float(i);
  }

  for(i=0; i<n; i++){
    printf("%f\n", ah[i]);
  }

  hipMalloc((void**)&ad, nbytes);
  hipMalloc((void**)&bd, nbytes);

  // ! first dest, than source !
  hipMemcpy(ad, ah, nbytes, hipMemcpyHostToDevice);
  hipMemcpy(bd, ad, nbytes, hipMemcpyDeviceToDevice);
  hipMemcpy(bh, bd, nbytes, hipMemcpyDeviceToHost);

  for(i=0; i<n; i++){
    printf("%f\n", bh[i]);
  }

  free(ah);
  free(bh);
  hipFree(ad);
  hipFree(bd);

  return 0;
}