#include "hip/hip_runtime.h"
#include "gpu_conv.h"

#ifdef __cplusplus
extern "C" {
#endif

void evaluate_convolution(tensor *m, tensor *h, conv_data *conv, param *p){

  for (int i=0; i<3; i++)
    if (p->demagCoarse[i]>1){
      fprintf(stderr, "abort: convolution on a coarse grid not yet implemented.\n");
      abort();
    }

  switch (p->kernelType){
    case KERNEL_MICROMAG3D:
      if (p->size[X]/p->demagCoarse[X] > 1)
        evaluate_micromag3d_conv(m, h, conv);
      if (p->size[X]/p->demagCoarse[X] == 1)
        evaluate_micromag3d_conv_Xthickness_1(m, h, conv);
      break;
    case KERNEL_MICROMAG2D:
      evaluate_micromag2d_conv(m, h, conv);
      break;
    default:
      fprintf(stderr, "abort: no valid kernelType\n");
      abort();
  }

  return;
}



void evaluate_micromag3d_conv(tensor *m, tensor *h, conv_data *conv){

  int m_length = m->len/3;
  int N = conv->fft1->len/3;

  float *m_comp[3], *h_comp[3], *fft1_comp[3];
  for (int i=0; i<3; i++){
    fft1_comp[i] = &conv->fft1->list[i*N];
    m_comp[i]    = &m->list[i*m_length];
    h_comp[i]    = &h->list[i*m_length];
  }

  float *fftMx = &conv->fft1->list[0*N];
  float *fftMy = &conv->fft1->list[1*N];
  float *fftMz = &conv->fft1->list[2*N];
  float *fftKxx = &conv->kernel->list[0*N/2];
  float *fftKxy = &conv->kernel->list[1*N/2];
  float *fftKxz = &conv->kernel->list[2*N/2];
  float *fftKyy = &conv->kernel->list[3*N/2];
  float *fftKyz = &conv->kernel->list[4*N/2];
  float *fftKzz = &conv->kernel->list[5*N/2];

    //Fourier transforming of fft_mi
  for(int i=0; i<3; i++)
    gpuFFT3dPlan_forward(conv->fftplan, m_comp[i], fft1_comp[i]);  ///@todo out-of-place
  
    // kernel multiplication
    gpu_kernelmul6(fftMx, fftMy, fftMz, fftKxx, fftKyy, fftKzz, fftKyz, fftKxz, fftKxy, N);

    //inverse Fourier transforming fft_hi
  for(int i=0; i<3; i++)
    gpuFFT3dPlan_inverse(conv->fftplan, fft1_comp[i], h_comp[i]);  ///@todo out-of-place

  return;
}

void evaluate_micromag3d_conv_Xthickness_1(tensor *m, tensor *h, conv_data *conv){

  int m_length = m->len/3;
  int N = conv->fft1->len/3;

  float *m_comp[3], *h_comp[3], *fft1_comp[3];
  for (int i=0; i<3; i++){
    fft1_comp[i] = &conv->fft1->list[i*N];
    m_comp[i]    = &m->list[i*m_length];
    h_comp[i]    = &h->list[i*m_length];
  }
  
  float *fftMx = &conv->fft1->list[0*N];
  float *fftMy = &conv->fft1->list[1*N];
  float *fftMz = &conv->fft1->list[2*N];
  float *fftKxx = &conv->kernel->list[0*N/2];
  float *fftKyy = &conv->kernel->list[1*N/2];
  float *fftKyz = &conv->kernel->list[2*N/2];
  float *fftKzz = &conv->kernel->list[3*N/2];


  //Fourier transforming of fft_mi
  for(int i=0; i<3; i++)
    gpuFFT3dPlan_forward(conv->fftplan, m_comp[i], fft1_comp[i]);  ///@todo out-of-place

    // kernel multiplication
  gpu_kernelmul4(fftMx, fftMy,  fftMz, fftKxx, fftKyy, fftKyz, fftKzz, N);

    //inverse Fourier transforming fft_hi
  for(int i=0; i<3; i++)
    gpuFFT3dPlan_inverse(conv->fftplan, fft1_comp[i], h_comp[i]);  ///@todo out-of-place


  return;
}

void evaluate_micromag2d_conv(tensor *m, tensor *h, conv_data *conv){

  int m_length = m->len/3;
  int N = conv->fft1->len/2;    // only 2 components need to be convolved!

  float *m_comp[2], *h_comp[2], *fft1_comp[2];
  for (int i=1; i<3; i++){
    fft1_comp[i-1] = &conv->fft1->list[(i-1)*N];
    m_comp[i-1]    = &m->list[i*m_length];
    h_comp[i-1]    = &h->list[i*m_length];
  }

  float *fftMy = &conv->fft1->list[0*N];
  float *fftMz = &conv->fft1->list[1*N];
  float *fftKyy = &conv->kernel->list[0*N/2];
  float *fftKyz = &conv->kernel->list[1*N/2];
  float *fftKzz = &conv->kernel->list[2*N/2];

    //Fourier transforming of fft_mi
  for(int i=0; i<2; i++)
    gpuFFT3dPlan_forward(conv->fftplan, m_comp[i], fft1_comp[i]);  ///@todo out-of-place

    // kernel multiplication
  gpu_kernelmul3(fftMy,  fftMz, fftKyy, fftKyz, fftKzz, N);

    //inverse Fourier transforming fft_hi
  for(int i=0; i<2; i++)
    gpuFFT3dPlan_inverse(conv->fftplan, fft1_comp[i], h_comp[i]);  ///@todo out-of-place

  return;
}


// ****************************************************************************************************



// functions for copying to and from padded matrix ****************************************************
conv_data *new_conv_data(param *p, tensor *kernel){

  ///@todo add a test that checks if the kernel has been initialized.   
  conv_data *conv = (conv_data *) calloc(1, sizeof(conv));
  int size4d[4] = {0, p->kernelSize[X], p->kernelSize[Y], gpu_pad_to_stride(p->kernelSize[Z]+2)};
  
  switch (p->kernelType){
    case KERNEL_MICROMAG3D:
      size4d[0] = 3;
      break;
    case KERNEL_MICROMAG2D:
      size4d[0] = 2;
      break;
    default:
      fprintf(stderr, "abort: no valid kernelType\n");
      abort();
  }

  conv->fft1 = new_gputensor(4, size4d);
  conv->fft2 = conv->fft1;
  conv->fftplan = new_gpuFFT3dPlan_padded(p->size, p->kernelSize);
  conv->kernel = kernel;

  return (conv);
}
// ****************************************************************************************************


// to be placed in gpu_kernmul.cu

__global__ void _gpu_kernelmul4(float* fftMx,  float* fftMy,  float* fftMz,
                                float* fftKxx, float* fftKyy, float* fftKyz, float* fftKzz, int N){
  int i = threadindex;
  int e = 2 * i;

  // we some shared memory here, which saves an "8N" buffer in the global memory
  ///@todo coalescale read/writes, cleanup indices
  if(i < N){
  float reMx = fftMx[e  ];
  float imMx = fftMx[e+1];

  float reMy = fftMy[e  ];
  float imMy = fftMy[e+1];

  float reMz = fftMz[e  ];
  float imMz = fftMz[e+1];

  float Kxx = fftKxx[i];
  float Kyy = fftKyy[i];
  float Kyz = fftKyz[i];
  float Kzz = fftKzz[i];
  
  fftMx[e  ] = reMx * Kxx;
  fftMx[e+1] = imMx * Kxx;
  fftMy[e  ] = reMy * Kyy + reMz * Kyz;
  fftMy[e+1] = imMy * Kyy + imMz * Kyz;
  fftMz[e  ] = reMy * Kyz + reMz * Kzz;
  fftMz[e+1] = imMy * Kyz + imMz * Kzz;
  }
  
  return;
}

void gpu_kernelmul4(float *fftMx, float *fftMy, float *fftMz, float *fftKxx, float *fftKyy, float *fftKyz, float *fftKzz, int nRealNumbers){

  timer_start("kernel_mul");
  assert(nRealNumbers > 0);
  assert(nRealNumbers % 2 == 0);

  dim3 gridSize, blockSize;
  make1dconf(nRealNumbers/2, &gridSize, &blockSize);

  _gpu_kernelmul4<<<gridSize, blockSize>>>(fftMx, fftMy, fftMz, fftKxx, fftKyy, fftKyz, fftKzz, nRealNumbers/2);
  gpu_sync();
  timer_stop("kernel_mul");
 
  return;
}


__global__ void _gpu_kernelmul3(float* fftMy,  float* fftMz,
                                float* fftKyy, float* fftKyz, float* fftKzz, int N){
  int i = threadindex;
  int e = 2 * i;

  // we some shared memory here, which saves an "8N" buffer in the global memory
  ///@todo coalescale read/writes, cleanup indices
  if(i < N){

  float reMy = fftMy[e  ];
  float imMy = fftMy[e+1];

  float reMz = fftMz[e  ];
  float imMz = fftMz[e+1];

  float Kyy = fftKyy[i];
  float Kyz = fftKyz[i];
  float Kzz = fftKzz[i];
  
  fftMy[e  ] = reMy * Kyy + reMz * Kyz;
  fftMy[e+1] = imMy * Kyy + imMz * Kyz;
  fftMz[e  ] = reMy * Kyz + reMz * Kzz;
  fftMz[e+1] = imMy * Kyz + imMz * Kzz;
  }
  
  return;
}

void gpu_kernelmul3(float *fftMy, float *fftMz, float *fftKyy, float *fftKyz, float *fftKzz, int nRealNumbers){

  timer_start("kernel_mul");
  assert(nRealNumbers > 0);
  assert(nRealNumbers % 2 == 0);

  dim3 gridSize, blockSize;
  make1dconf(nRealNumbers/2, &gridSize, &blockSize);

  _gpu_kernelmul3<<<gridSize, blockSize>>>(fftMy, fftMz, fftKyy, fftKyz, fftKzz, nRealNumbers/2);
  gpu_sync();
  timer_stop("kernel_mul");
 
  return;
}






#ifdef __cplusplus
}
#endif