#include "hip/hip_runtime.h"
#include "gpunormalize.h"

#ifdef __cplusplus
extern "C" {
#endif


__global__ void _gpu_normalize(float* mx , float* my , float* mz){
  int i = ((blockIdx.x * blockDim.x) + threadIdx.x);
  float norm = rsqrtf(mx[i]*mx[i] + my[i]*my[i] + mz[i]*mz[i]);     // inverse square root
  mx[i] *= norm;
  my[i] *= norm;
  mz[i] *= norm;
}


__global__ void _gpu_normalize_map(float* mx , float* my , float* mz, float* normMap){
  int i = ((blockIdx.x * blockDim.x) + threadIdx.x);
  float norm = rsqrtf(mx[i]*mx[i] + my[i]*my[i] + mz[i]*mz[i]) * normMap[i];
  mx[i] *= norm;
  my[i] *= norm;
  mz[i] *= norm;
}


void gpu_normalize_uniform(float* m, int N){

  int gridSize = -1, blockSize = -1;
  make1dconf(N, &gridSize, &blockSize);

  float* mx = &(m[0*N]);
  float* my = &(m[1*N]);
  float* mz = &(m[2*N]);

//   timer_start("normalize");
  _gpu_normalize<<<gridSize, blockSize>>>(mx, my, mz);
  gpu_sync();
//   timer_stop("normalize");
}

void gpu_normalize_map(float* m, float* map, int N){

  int gridSize = -1, blockSize = -1;
  make1dconf(N, &gridSize, &blockSize);

  float* mx = &(m[0*N]);
  float* my = &(m[1*N]);
  float* mz = &(m[2*N]);

//   timer_start("normalize");
  _gpu_normalize_map<<<gridSize, blockSize>>>(mx, my, mz, map);
  gpu_sync();
//   timer_stop("normalize");
  
}


void gpu_normalize(param* p, tensor* m){

  int complen = m->len / 3;

  int gridSize = -1, blockSize = -1;
  make1dconf(complen, &gridSize, &blockSize);

  float* mx = &(m->list[0*complen]);
  float* my = &(m->list[1*complen]);
  float* mz = &(m->list[2*complen]);

//   timer_start("normalize");
  if(p->msatMap == NULL){
    _gpu_normalize<<<gridSize, blockSize>>>(mx, my, mz);
  }
  else{
    _gpu_normalize_map<<<gridSize, blockSize>>>(mx, my, mz, p->msatMap->list);
  }
  gpu_sync();
//   timer_stop("normalize");
}



#ifdef __cplusplus
}
#endif