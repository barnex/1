#include "hip/hip_runtime.h"
#include "gpunormalize.h"

#ifdef __cplusplus
extern "C" {
#endif


__global__ void _gpu_normalize(float* mx , float* my , float* mz, int N){
  
  int i = threadindex;
  if (i<N){
  
  float norm = rsqrtf(mx[i]*mx[i] + my[i]*my[i] + mz[i]*mz[i]);     // inverse square root
  mx[i] *= norm;
  my[i] *= norm;
  mz[i] *= norm;
  }
  return;
}


__global__ void _gpu_normalize_map(float* mx , float* my , float* mz, float* normMap, int N){
  
  int i = threadindex;
  
  if (i<N){
  float norm = rsqrtf(mx[i]*mx[i] + my[i]*my[i] + mz[i]*mz[i]) * normMap[i];
  mx[i] *= norm;
  my[i] *= norm;
  mz[i] *= norm;
  }
  
  return;
}


void gpu_normalize_uniform(float* m, int N){

  dim3 gridSize, blockSize;
  make1dconf(N, &gridSize, &blockSize);

  float* mx = &(m[0*N]);
  float* my = &(m[1*N]);
  float* mz = &(m[2*N]);

//   timer_start("normalize");
  _gpu_normalize<<<gridSize, blockSize>>>(mx, my, mz, N);
  gpu_sync();
//   timer_stop("normalize");
}

void gpu_normalize_map(float* m, float* map, int N){

  dim3 gridSize, blockSize;
  make1dconf(N, &gridSize, &blockSize);

  float* mx = &(m[0*N]);
  float* my = &(m[1*N]);
  float* mz = &(m[2*N]);

//   timer_start("normalize");
  _gpu_normalize_map<<<gridSize, blockSize>>>(mx, my, mz, map, N);
  gpu_sync();
//   timer_stop("normalize");
  
}


void gpu_normalize(param* p, tensor* m){

  int N = m->len / 3;
  dim3 gridSize, blockSize;
  make1dconf(N, &gridSize, &blockSize);

  float* mx = &(m->list[0*N]);
  float* my = &(m->list[1*N]);
  float* mz = &(m->list[2*N]);

//   timer_start("normalize");
  if(p->msatMap == NULL)
    _gpu_normalize<<<gridSize, blockSize>>>(mx, my, mz, N);
  else
    _gpu_normalize_map<<<gridSize, blockSize>>>(mx, my, mz, p->msatMap->list, N);

  gpu_sync();
//   timer_stop("normalize");
}



#ifdef __cplusplus
}
#endif