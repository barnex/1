#include "gpu_safe.h"
#include <stdio.h>
#include <hipfft/hipfft.h>

#ifdef __cplusplus
extern "C" {
#endif


// void gpu_safe(int status){
//   if(status != hipSuccess){
//     fprintf(stderr, "received CUDA error: %s\n", hipGetErrorString((hipError_t)status));
//     abort();
//   }
// }


char* cufftGetErrorString(hipfftResult s){
  switch(s){
    default: return "Unknown error";
    case HIPFFT_SUCCESS: return "Any CUFFT operation is successful.";
    case HIPFFT_INVALID_PLAN: return "CUFFT is passed an invalid plan handle.";
    case HIPFFT_ALLOC_FAILED: return "CUFFT failed to allocate GPU memory.";
    case HIPFFT_INVALID_TYPE: return "The user requests an unsupported type.";
    case HIPFFT_INVALID_VALUE: return "The user specifies a bad memory pointer.";
    case HIPFFT_INTERNAL_ERROR: return "Used for all internal driver errors.";
    case HIPFFT_EXEC_FAILED: return "CUFFT failed to execute an FFT on the GPU.";
    case HIPFFT_SETUP_FAILED: return "The CUFFT library failed to initialize.";
//     case CUFFT_SHUTDOWN_FAILED: return "The CUFFT library failed to shut down.";
    case HIPFFT_INVALID_SIZE: return "The user specifies an unsupported FFT size.";
  }
}

#ifdef __cplusplus
}
#endif
