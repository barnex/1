#include "gputil.h"
#include <hipfft/hipfft.h>
#include "gpu_transpose.h"
#include "gpu_transpose2.h"
#include "gpu_safe.h"
#include "gpu_fft4.h"
#include "gpu_fft5.h"
#include "gpu_fftbig.h"
#include "gpu_conf.h"

#ifdef __cplusplus
extern "C" {
#endif

gpuFFT3dPlan_big* new_gpuFFT3dPlan_padded_big(int* size, int* paddedSize){

  int N0 = size[X];
  int N1 = size[Y];
  int N2 = size[Z];
  
  assert(paddedSize[X] > 0);
  assert(paddedSize[Y] > 1);
  assert(paddedSize[Z] > 1);
  
  gpuFFT3dPlan_big* plan = (gpuFFT3dPlan_big*)malloc(sizeof(gpuFFT3dPlan_big));
  
  plan->size = (int*)calloc(3, sizeof(int));    ///@todo not int* but int[3]
  plan->paddedSize = (int*)calloc(3, sizeof(int));
  plan->paddedStorageSize = (int*)calloc(3, sizeof(int));
  
//   int* paddedSize = plan->paddedSize;
  int* paddedStorageSize = plan->paddedStorageSize;
  
  plan->size[0] = N0; 
  plan->size[1] = N1; 
  plan->size[2] = N2;
  plan->N = N0 * N1 * N2;
  
  plan->paddedSize[X] = paddedSize[X];
  plan->paddedSize[Y] = paddedSize[Y];
  plan->paddedSize[Z] = paddedSize[Z];
  plan->paddedN = plan->paddedSize[0] * plan->paddedSize[1] * plan->paddedSize[2];
  
  plan->paddedStorageSize[X] = plan->paddedSize[X];
  plan->paddedStorageSize[Y] = plan->paddedSize[Y];
  plan->paddedStorageSize[Z] = plan->paddedSize[Z] + 2;
  plan->paddedStorageN = paddedStorageSize[X] * paddedStorageSize[Y] * paddedStorageSize[Z];

  plan->fwPlanZ = (bigfft *) malloc(sizeof(bigfft));
  plan->invPlanZ = (bigfft *) malloc(sizeof(bigfft));
  plan->planY = (bigfft *) malloc(sizeof(bigfft));
  plan->planX = (bigfft *) malloc(sizeof(bigfft));
  if ( paddedSize[X]!=size[X] || paddedSize[Y]!=size[Y]){
    init_bigfft(plan->fwPlanZ , paddedSize[Z], plan->paddedStorageSize[Z], HIPFFT_R2C, size[X]*size[Y]);
    init_bigfft(plan->invPlanZ, plan->paddedStorageSize[Z], paddedSize[Z], HIPFFT_C2R, size[X]*size[Y]);
  }
  else{
    init_bigfft(plan->fwPlanZ , plan->paddedStorageSize[Z], plan->paddedStorageSize[Z], HIPFFT_R2C, size[X]*size[Y]);
    init_bigfft(plan->invPlanZ, plan->paddedStorageSize[Z], plan->paddedStorageSize[Z], HIPFFT_C2R, size[X]*size[Y]);
  }
/*  init_bigfft(plan->planY, 2*plan->paddedSize[Y], 2*plan->paddedSize[Y], HIPFFT_C2C, paddedStorageSize[Z] * size[X] / 2);
  init_bigfft(plan->planX, 2*plan->paddedSize[X], 2*plan->paddedSize[X], HIPFFT_C2C, paddedStorageSize[Z] * paddedSize[Y] / 2);*/
  init_bigfft(plan->planY, plan->paddedSize[Y], plan->paddedSize[Y], HIPFFT_C2C, paddedStorageSize[Z] * size[X] / 2);
  init_bigfft(plan->planX, plan->paddedSize[X], plan->paddedSize[X], HIPFFT_C2C, paddedStorageSize[Z] * paddedSize[Y] / 2);
  
  
  for (int i=0; i<plan->fwPlanZ->Nbatch; i++)
    printf("fwZ: Nbatch: %d, i: %d, batch: %d, batch_index_in: %d, batch_index_out: %d\n", plan->fwPlanZ->Nbatch, i, plan->fwPlanZ->batch[i], plan->fwPlanZ->batch_index_in[i], plan->fwPlanZ->batch_index_out[i]);
  printf("\n");  
  for (int i=0; i<plan->invPlanZ->Nbatch; i++)
    printf("invZ: Nbatch: %d, i: %d, batch: %d, batch_index_in: %d, batch_index_out: %d\n", plan->invPlanZ->Nbatch, i, plan->invPlanZ->batch[i], plan->invPlanZ->batch_index_in[i], plan->invPlanZ->batch_index_out[i]);
  printf("\n");  
  for (int i=0; i<plan->planY->Nbatch; i++)
    printf("Y: Nbatch: %d, i: %d, batch: %d, batch_index_in: %d, batch_index_out: %d\n", plan->planY->Nbatch, i, plan->planY->batch[i], plan->planY->batch_index_in[i], plan->planY->batch_index_out[i]);
  printf("\n");  
  for (int i=0; i<plan->planX->Nbatch; i++)
    printf("X: Nbatch: %d, i: %d, batch: %d, batch_index_in: %d, batch_index_out: %d\n", plan->planX->Nbatch, i, plan->planX->batch[i], plan->planX->batch_index_in[i], plan->planX->batch_index_out[i]);
  printf("\n");  
/*  

  
  plan->Nbatch    = (int *) calloc(3, sizeof(int));
  plan->batch     = (int**) calloc(3, sizeof(int*));
  plan->batch_cum = (int**) calloc(3, sizeof(int*));

  int *Nbatch = plan->Nbatch;
  int **batch = plan->batch;

  init_batch_fft_big(plan, Z, size[X]*size[Y], plan->paddedStorageSize[Z]);
  for (int i=0; i<Nbatch[Z]; i++)
    printf("Z: Nbatch: %d, i: %d, batch: %d, el. in batch: %d, %d, batch_cum: %d\n", Nbatch[Z], i, batch[Z][i], batch[Z][i]*paddedSize[Z], batch[Z][i]*paddedStorageSize[Z], plan->batch_cum[Z][i]);
  printf("\n\n");
  init_batch_fft_big(plan, Y, paddedStorageSize[Z] * size[X] / 2, 2*plan->paddedSize[Y]);   //2* because we need the number of floats
  for (int i=0; i<Nbatch[Y]; i++)
    printf("Y: Nbatch: %d, i: %d, batch: %d, el. in batch: %d, batch_cum: %d\n", Nbatch[Y], i, batch[Y][i], batch[Y][i]*plan->paddedStorageSize[Y], plan->batch_cum[Y][i]);
  printf("\n\n");
  init_batch_fft_big(plan, X, paddedStorageSize[Z] * paddedSize[Y] / 2, 2*plan->paddedSize[X]);
  for (int i=0; i<Nbatch[X]; i++)
    printf("X: Nbatch: %d, i: %d, batch: %d, el. in batch: %d, batch_cum: %d\n", Nbatch[X], i, batch[X][i], batch[X][i]*plan->paddedStorageSize[X], plan->batch_cum[X][i]);
  printf("\n\n");
 
  // plan assignment for batch PlanZ -------------------------------------
  gpu_safefft( hipfftPlan1d( &plan->fwPlanZ_1, plan->paddedSize[Z], HIPFFT_R2C, batch[Z][0]) );
  gpu_safefft( hipfftPlan1d( &plan->invPlanZ_1, plan->paddedSize[Z], HIPFFT_C2R, batch[Z][0]) );
  if ( batch[Z][Nbatch[Z]-1] != batch[Z][0] ){
    gpu_safefft( hipfftPlan1d( &plan->fwPlanZ_2, plan->paddedSize[Z], HIPFFT_R2C, batch[Z][Nbatch[Z]-1]) );
    gpu_safefft( hipfftPlan1d( &plan->invPlanZ_2, plan->paddedSize[Z], HIPFFT_C2R, batch[Z][Nbatch[Z]-1]) );
  }

  plan->fwPlanZ  = (hipfftHandle *) calloc(Nbatch[Z], sizeof(hipfftHandle));
  plan->invPlanZ = (hipfftHandle *) calloc(Nbatch[Z], sizeof(hipfftHandle));
  
  for (int i=0; i<Nbatch[Z]; i++)
    if ( batch[Z][i] == batch[Z][0] ){
      plan->fwPlanZ[i] = plan->fwPlanZ_1;
      plan->invPlanZ[i] = plan->invPlanZ_1;
    }
    else{
      plan->fwPlanZ[i] = plan->fwPlanZ_2;
      plan->invPlanZ[i] = plan->invPlanZ_2;
    }
  //-----------------------------------------------------------------------
 

  // plan assignment for batch PlanY -------------------------------------
  gpu_safefft( hipfftPlan1d( &plan->PlanY_1, plan->paddedSize[Y], HIPFFT_C2C, batch[Y][0]) );
  if ( batch[Y][Nbatch[Y]-1] != batch[Y][0] ){
    gpu_safefft( hipfftPlan1d( &plan->PlanY_2, plan->paddedSize[Y], HIPFFT_C2C, batch[Y][Nbatch[Y]-1]) );
  }

  plan->planY  = (hipfftHandle *) calloc(Nbatch[Y], sizeof(hipfftHandle));
  
  for (int i=0; i<Nbatch[Y]; i++)
    if ( batch[Y][i] == batch[Y][0] )
      plan->planY[i] = plan->PlanY_1;
    else
      plan->planY[i] = plan->PlanY_2;

  //-----------------------------------------------------------------------


  // plan assignment for batch PlanX -------------------------------------
  gpu_safefft( hipfftPlan1d( &plan->PlanX_1, plan->paddedSize[X], HIPFFT_C2C, batch[X][0]) );
  if ( batch[X][Nbatch[X]-1] != batch[X][0] ){
    gpu_safefft( hipfftPlan1d( &plan->PlanX_2, plan->paddedSize[X], HIPFFT_C2C, batch[X][Nbatch[X]-1]) );
  }

  plan->planX  = (hipfftHandle *) calloc(Nbatch[X], sizeof(hipfftHandle));
  
  for (int i=0; i<Nbatch[X]; i++)
    if ( batch[X][i] == batch[X][0] )
      plan->planX[i] = plan->PlanX_1;
    else
      plan->planX[i] = plan->PlanX_2;
  //-----------------------------------------------------------------------*/

//   gpu_safefft( hipfftPlan1d((&plan->PlanX_1), plan->paddedSize[X], HIPFFT_C2C, paddedStorageSize[Z] * paddedSize[Y] / 2) );
//   gpu_safefft( hipfftPlan1d((&plan->invPlanZ_1), plan->paddedSize[Z], HIPFFT_C2R, size[X]*size[Y]) );
  
  plan->transp = new_gpu_array(plan->paddedStorageN);
  
  return plan;
}




void gpuFFT3dPlan_forward_big(gpuFFT3dPlan_big* plan, float* input, float* output){
//   timer_start("gpu_plan3d_real_input_forward_exec");

  int* size = plan->size;
  int* pSSize = plan->paddedStorageSize;
  int N0 = pSSize[X];
  int N1 = pSSize[Y];
  int N2 = pSSize[Z]/2; // we treat the complex data as an N0 x N1 x N2 x 2 array
  int N3 = 2;
  
  int half_pSSize = plan->paddedStorageN/2;
  
  //     zero out the output matrix
    gpu_zero(output, plan->paddedStorageN);
  //     padding of the input matrix towards the output matrix
    gpu_copy_to_pad(input, output, size, pSSize);

  
//  float* data = input;
  float* data = output;
  float* data2 = plan->transp; 

  if ( pSSize[X]!=size[X] || pSSize[Y]!=size[Y]){
      //out of place FFTs in Z-direction from the 0-element towards second half of the zeropadded matrix (out of place: no +2 on input!)
   bigfft_execR2C(plan->fwPlanZ, (hipfftReal*)data,  (hipfftComplex*) (data + half_pSSize));                // it's in data
   gpu_sync();

      // zero out the input data points at the start of the matrix
    gpu_zero(data, size[X]*size[Y]*pSSize[Z]);
    
      // YZ-transpose within the same matrix from the second half of the matrix towards the 0-element
    yz_transpose_in_place_fw(data, size, pSSize);                                                          // it's in data

      // in place FFTs in Y-direction
    bigfft_execC2C(plan->planY, (hipfftComplex*)data,  (hipfftComplex*)data, HIPFFT_FORWARD);                 // it's in data
    gpu_sync();
  }
  
  else {          //no zero padding in X- and Y direction (e.g. for Greens kernel computations)
      // in place FFTs in Z-direction (there is no zero space to perform them out of place)
    bigfft_execR2C(plan->fwPlanZ, (hipfftReal*)data,  (hipfftComplex*) (data));                              // it's in data
    gpu_sync();
    
      // YZ-transpose needs to be out of place.
    gpu_transposeYZ_complex(data, data2, N0, N1, N2*N3);                                                   // it's in data2
    
      // perform the FFTs in the Y-direction
    bigfft_execC2C(plan->planY, (hipfftComplex*)data2,  (hipfftComplex*)data, HIPFFT_FORWARD);                // it's in data
    gpu_sync();
  }

  if(N0 > 1){    // not done for 2D transforms
      // XZ transpose still needs to be out of place
    gpu_transposeXZ_complex(data, data2, N0, N2, N1*N3);                                                   // it's in data2
 
    
      // out of place FFTs in X-direction
    bigfft_execC2C(plan->planX, (hipfftComplex*)data2, (hipfftComplex*)output, HIPFFT_FORWARD);               // it's in output
    gpu_sync();
    
  }

//   timer_stop("gpu_plan3d_real_input_forward_exec");
  
  return;
}




void gpuFFT3dPlan_inverse_big(gpuFFT3dPlan_big* plan, float* input, float* output){
  
//   timer_start("gpu_plan3d_real_input_inverse_exec");
//   printf("start inverse\n");
  int* size = plan->size;
  int* pSSize = plan->paddedStorageSize;
  int N0 = pSSize[X];
  int N1 = pSSize[Y];
  int N2 = pSSize[Z]/2; // we treat the complex data as an N0 x N1 x N2 x 2 array
  int N3 = 2;
  int half_pSSize = plan->paddedStorageN/2;
  
  float* data = input;
  float* data2 = plan->transp; // both the transpose and FFT are out-of-place between data and data2

  if (N0 > 1){
      // out of place FFTs in the X-direction (i.e. no +2 stride on input!)
    bigfft_execC2C(plan->planX, (hipfftComplex*)data,  (hipfftComplex*)data2, HIPFFT_BACKWARD);                // it's in data2
    gpu_sync();

      // XZ transpose still needs to be out of place
    gpu_transposeXZ_complex(data2, data, N1, N2, N0*N3);                                                   // it's in data
  }
  
  if ( pSSize[X]!=size[X] || pSSize[Y]!=size[Y]){
      // in place FFTs in Y-direction
    bigfft_execC2C(plan->planY, (hipfftComplex*)data,  (hipfftComplex*)data, HIPFFT_BACKWARD);                 // it's in data
    gpu_sync();
    
      // YZ-transpose within the same matrix from the 0-element towards the second half of the matrix
    yz_transpose_in_place_inv(data, size, pSSize);                                                         // it's in data

      // out of place FFTs in Z-direction from the second half of the matrix towards the 0-element
    bigfft_execC2R(plan->invPlanZ, (hipfftComplex*)(data + half_pSSize), (hipfftReal*)data );                // it's in data
    gpu_sync();

  }
  else {          //no zero padding in X- and Y direction (e.g. for Greens kernel computations)
      // out of place FFTs in Y-direction
    bigfft_execC2C(plan->planY, (hipfftComplex*)data,  (hipfftComplex*)data2, HIPFFT_BACKWARD);                // it's in data2
    gpu_sync();
    
      // YZ-transpose needs to be out of place.
    gpu_transposeYZ_complex(data2, data, N0, N2, N1*N3);                                                   // it's in data   

      // in place FFTs in Z-direction
    bigfft_execC2R(plan->invPlanZ, (hipfftComplex*) data, (hipfftReal*) data );                              // it's in data
    gpu_sync();
  }
  
  gpu_copy_to_unpad(data, output, pSSize, size);                                                           // it's in output
 
//   timer_stop("gpu_plan3d_real_input_inverse_exec");
  
  return;
}



#ifdef __cplusplus
}
#endif