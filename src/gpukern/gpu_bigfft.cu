#include "gpu_bigfft.h"
#include "gpu_safe.h"

#ifdef __cplusplus
extern "C" {
#endif

void init_bigfftR2C(bigfft* target, int size, int batch){
  gpu_safefft( hipfftPlan1d(&(target->plan), size, HIPFFT_R2C, batch) );
  gpu_safefft( cufftSetCompatibilityMode((target->plan), CUFFT_COMPATIBILITY_NATIVE) );
}

void init_bigfftC2R(bigfft* target, int size, int batch){
  gpu_safefft( hipfftPlan1d(&(target->plan), size, HIPFFT_C2R, batch) );
  gpu_safefft( cufftSetCompatibilityMode((target->plan), CUFFT_COMPATIBILITY_NATIVE) );
}

void init_bigfftC2C(bigfft* target, int size, int batch){
  gpu_safefft( hipfftPlan1d(&(target->plan), size, HIPFFT_C2C, batch) );
  gpu_safefft( cufftSetCompatibilityMode((target->plan), CUFFT_COMPATIBILITY_NATIVE) );
}

void bigfft_execR2C(bigfft* plan, float* input, float* output){
  gpu_safefft( hipfftExecR2C(plan->plan, (hipfftReal*)input, (hipfftComplex*)output) );
}

void bigfft_execC2R(bigfft* plan, float* input, float* output){
  gpu_safefft( hipfftExecC2R(plan->plan, (hipfftComplex*)input, (hipfftReal*)output) );
}

void bigfft_execC2C(bigfft* plan, float* input, float* output, int direction){
  gpu_safefft( hipfftExecC2C(plan->plan, (hipfftComplex*)input, (hipfftComplex*)output, direction) );
}


#ifdef __cplusplus
}
#endif
