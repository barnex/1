#include "gpu_bigfft.h"
#include "gpu_safe.h"

#ifdef __cplusplus
extern "C" {
#endif

void init_bigfft(bigfft* target, int size, hipfftType type, int batch){
  gpu_safefft( hipfftPlan1d(&(target->plan), size, type, batch) );
  gpu_safefft( cufftSetCompatibilityMode((target->plan), CUFFT_COMPATIBILITY_NATIVE) );
}


void bigfft_execR2C(bigfft* plan, hipfftReal* input, hipfftComplex* output){
  gpu_safefft( hipfftExecR2C(plan->plan, (hipfftReal*)input, (hipfftComplex*)output) );
}

void bigfft_execC2R(bigfft* plan, hipfftComplex* input, hipfftReal* output){
  gpu_safefft( hipfftExecC2R(plan->plan, (hipfftComplex*)input, (hipfftReal*)output) );
}

void bigfft_execC2C(bigfft* plan, hipfftComplex* input, hipfftComplex* output, int direction){
  gpu_safefft( hipfftExecC2C(plan->plan, (hipfftComplex*)input, (hipfftComplex*)output, direction) );
}


#ifdef __cplusplus
}
#endif
