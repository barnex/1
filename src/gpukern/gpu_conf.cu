#include "hip/hip_runtime.h"
/*
 *  This file is part of MuMax, a high-performance micromagnetic simulator.
 *  Copyright 2010  Arne Vansteenkiste, Ben Van de Wiele.
 *  Use of this source code is governed by the GNU General Public License version 3
 *  (as published by the Free Software Foundation) that can be found in the license.txt file.
 *
 *  Note that you are welcome to modify this code under condition that you do not remove any 
 *  copyright notices and prominently state that you modified it, giving a relevant date.
 */

#include "gpu_conf.h"
#include "gpu_properties.h"
#include "../macros.h"
#include <assert.h>

#ifdef __cplusplus
extern "C" {
#endif

void check3dconf(dim3 gridSize, dim3 blockSize){

  debugvv( printf("check3dconf((%d, %d, %d),(%d, %d, %d))\n", gridSize.x, gridSize.y, gridSize.z, blockSize.x, blockSize.y, blockSize.z) );
  
  hipDeviceProp_t* prop = (hipDeviceProp_t*)gpu_getproperties();
  int maxThreadsPerBlock = prop->maxThreadsPerBlock;
  int* maxBlockSize = prop->maxThreadsDim;
  int* maxGridSize = prop->maxGridSize;
  
  assert(gridSize.x > 0);
  assert(gridSize.y > 0);
  assert(gridSize.z > 0);
  
  assert(blockSize.x > 0);
  assert(blockSize.y > 0);
  assert(blockSize.z > 0);
  
  assert(blockSize.x <= maxBlockSize[X]);
  assert(blockSize.y <= maxBlockSize[Y]);
  assert(blockSize.z <= maxBlockSize[Z]);
  
  assert(gridSize.x <= maxGridSize[X]);
  assert(gridSize.y <= maxGridSize[Y]);
  assert(gridSize.z <= maxGridSize[Z]);
  
  assert(blockSize.x * blockSize.y * blockSize.z <= maxThreadsPerBlock);
}

void check1dconf(int gridsize, int blocksize){
  assert(gridsize > 0);
  assert(blocksize > 0);
  assert(blocksize <= ((hipDeviceProp_t*)gpu_getproperties())->maxThreadsPerBlock);
}

int _gpu_max_threads_per_block = 0;

int gpu_maxthreads(){
  if(_gpu_max_threads_per_block <= 0){
    hipDeviceProp_t* prop = (hipDeviceProp_t*)gpu_getproperties();
    _gpu_max_threads_per_block = prop->maxThreadsPerBlock;
  }
  return _gpu_max_threads_per_block;
}

void gpu_setmaxthreads(int max){
  _gpu_max_threads_per_block = max;
}

void make1dconf(int N, dim3* gridSize, dim3* blockSize){

//   debugvv( printf("make1dconf(%d)\n", N) );
  
  hipDeviceProp_t* prop = (hipDeviceProp_t*)gpu_getproperties();
  int maxBlockSize = gpu_maxthreads();
//   if(maxBlockSize > 128){
//     fprintf(stderr, "WARNING: using 128 as max block size! \n");
//     maxBlockSize = 128;
//   }
  int maxGridSize = prop->maxGridSize[X];

  (*blockSize).x = maxBlockSize;
  (*blockSize).y = 1;
  (*blockSize).z = 1;
  
  int N2 = divUp(N, maxBlockSize); // N2 blocks left
  
  int NX = divUp(N2, maxGridSize);
  int NY = divUp(N2, NX);

  (*gridSize).x = NX;
  (*gridSize).y = NY;
  (*gridSize).z = 1;

  assert((*gridSize).x * (*gridSize).y * (*gridSize).z * (*blockSize).x * (*blockSize).y * (*blockSize).z >= N);
  //assert((*gridSize).x * (*gridSize).y * (*gridSize).z * (*blockSize).x * (*blockSize).y * (*blockSize).z < N + maxBlockSize); ///@todo remove this assertion for very large problems
  
  check3dconf(*gridSize, *blockSize);
}

#ifdef __cplusplus
}
#endif
