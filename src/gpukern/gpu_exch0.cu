#include "hip/hip_runtime.h"
#include "gpu_exch.h"
#include "gpu_safe.h"
#include "../macros.h"

#ifdef __cplusplus
extern "C" {
#endif

/// @todo wrap
/// 2D, plane per plane, i=plane index
__global__ void _gpu_add_exch6(float* mx, float* my, float* mz,
                               float* hx, float* hy, float* hz,
                               int N0, int N1, int N2,
                               int wrap0, int wrap1, int wrap2,
							   float fac0, float fac1, float fac2, 
							   int i){

  //  i is passed
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int k = blockIdx.y * blockDim.y + threadIdx.y;
  int I = i*N1*N2 + j*N2 + k; // linear array index
  
  if (j < N1 && k < N2){

    // Local H initiated
	float Hx = hx[I];
	float Hy = hy[I];
	float Hz = hz[I];

	float mx1, my1, mz1; // magnetization of neighbor 1
	float mx2, my2, mz2; // magnetization of neighbor 2
	
	// Now add Neighbors.

    // neighbors in X direction
	int idx;
    if (i-1 >= 0){
      idx = (i-1)*N1*N2 + j*N2 + k;
    } else {
      idx = I;
    }
	mx1 = mx[idx]; my1 = my[idx]; mz1 = mz[idx];

 	if (i+1 < N0){
      idx = (i+1)*N1*N2 + j*N2 + k;
    } else {
      idx = I;
    } 
	mx2 = mx[idx]; my2 = my[idx]; mz2 = mz[idx];

    Hx += fac0 * (mx1 + mx2 - 2.0f*mx[I]);
    Hy += fac0 * (my1 + my2 - 2.0f*my[I]);
    Hz += fac0 * (mz1 + mz2 - 2.0f*mz[I]);

    // neighbors in Y direction
    if (j-1 >= 0){
      idx = (i)*N1*N2 + (j-1)*N2 + k;
    } else {
      idx = I;
    }
	mx1 = mx[idx]; my1 = my[idx]; mz1 = mz[idx];

 	if (j+1 < N1){
      idx = (i)*N1*N2 + (j+1)*N2 + k;
    } else {
      idx = I;
    } 
	mx2 = mx[idx]; my2 = my[idx]; mz2 = mz[idx];

    Hx += fac1 * (mx1 + mx2 - 2.0f*mx[I]);
    Hy += fac1 * (my1 + my2 - 2.0f*my[I]);
    Hz += fac1 * (mz1 + mz2 - 2.0f*mz[I]);

    // neighbors in Z direction
    if (k-1 >= 0){
      idx = (i)*N1*N2 + (j)*N2 + (k-1);
    } else {
      idx = I;
    }
	mx1 = mx[idx]; my1 = my[idx]; mz1 = mz[idx];

 	if (k+1 < N2){
      idx =  (i)*N1*N2 + (j)*N2 + (k+1);
    } else {
      idx = I;
    } 
	mx2 = mx[idx]; my2 = my[idx]; mz2 = mz[idx];
   
    Hx += fac2 * (mx1 + mx2 - 2.0f*mx[I]);
    Hy += fac2 * (my1 + my2 - 2.0f*my[I]);
    Hz += fac2 * (mz1 + mz2 - 2.0f*mz[I]);

	// Write back to global memory
    hx[I] = Hx;
    hy[I] = Hy;
    hz[I] = Hz;
  }
  
}


#define BLOCKSIZE 16
void gpu_add_exch(float* m, float* h, int N0, int N1, int N2, int wrap0, int wrap1, int wrap2, float cellsize0, float cellsize1, float cellsize2, int type){
  assert(type == 6);
  dim3 gridsize(divUp(N1, BLOCKSIZE), divUp(N2, BLOCKSIZE));
  dim3 blocksize(BLOCKSIZE, BLOCKSIZE, 1);
  int N = N0 * N1 * N2;

  float fac0 = 1.0f/(cellsize0 * cellsize0);
  float fac1 = 1.0f/(cellsize1 * cellsize1);
  float fac2 = 1.0f/(cellsize2 * cellsize2);

  for(int i=0; i<N0; i++){
    _gpu_add_exch6<<<gridsize, blocksize>>>(&m[0*N], &m[1*N], &m[2*N], &h[0*N], &h[1*N], &h[2*N], N0, N1, N2, wrap0, wrap1, wrap2, fac0, fac1, fac2, i);
  }
  gpu_sync();
}



#ifdef __cplusplus
}
#endif
