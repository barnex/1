#include "hip/hip_runtime.h"
#include "gpu_fft.h"
#include "../macros.h"
#include "gpu_transpose.h"
#include "gpu_safe.h"
#include "gpu_conf.h"
#include "gpu_mem.h"
// #include "//timer.h"
#include <stdio.h>
#include <assert.h>

#ifdef __cplusplus
extern "C" {
#endif


/**
 * Creates a new FFT plan for transforming the magnetization. 
 * Zero-padding in each dimension is optional, and rows with
 * only zero's are not transformed.
 * @todo on compute capability < 2.0, the first step is done serially...
 * @todo rename kernelsize -> paddedsize
 */
gpuFFT3dPlan* new_gpuFFT3dPlan_padded(int* size, int* paddedSize){
  
  int N0 = size[X];
  int N1 = size[Y];
  int N2 = size[Z];
  
  assert(paddedSize[X] > 0);
  assert(paddedSize[Y] > 1);
  assert(paddedSize[Z] > 1);
  
  gpuFFT3dPlan* plan = (gpuFFT3dPlan*)malloc(sizeof(gpuFFT3dPlan));
  
  plan->size = (int*)calloc(3, sizeof(int));    ///@todo not int* but int[3]
  plan->paddedSize = (int*)calloc(3, sizeof(int));
  plan->paddedStorageSize = (int*)calloc(3, sizeof(int));
  
//   int* paddedSize = plan->paddedSize;
  int* paddedStorageSize = plan->paddedStorageSize;
  
  plan->size[0] = N0; 
  plan->size[1] = N1; 
  plan->size[2] = N2;
  plan->N = N0 * N1 * N2;
  
  plan->paddedSize[X] = paddedSize[X];
  plan->paddedSize[Y] = paddedSize[Y];
  plan->paddedSize[Z] = paddedSize[Z];
  plan->paddedN = plan->paddedSize[0] * plan->paddedSize[1] * plan->paddedSize[2];
  
  plan->paddedStorageSize[X] = plan->paddedSize[X];
  plan->paddedStorageSize[Y] = plan->paddedSize[Y];
  plan->paddedStorageSize[Z] = gpu_pad_to_stride( plan->paddedSize[Z] + 2 );
  plan->paddedStorageN = paddedStorageSize[X] * paddedStorageSize[Y] * paddedStorageSize[Z];
  
  ///@todo check these sizes !
//  gpu_safe( hipfftPlan1d(&(plan->fwPlanZ), plan->paddedSize[Z], HIPFFT_R2C, 1) );
  gpu_safefft( hipfftPlan1d(&(plan->fwPlanZ), plan->paddedSize[Z], HIPFFT_R2C, size[Y]) );
  gpu_safefft( hipfftPlan1d(&(plan->planY), plan->paddedSize[Y], HIPFFT_C2C, paddedStorageSize[Z] * size[X] / 2) );          // IMPORTANT: the /2 is necessary because the complex transforms have only half the amount of elements (the elements are now complex numbers)
  gpu_safefft( hipfftPlan1d(&(plan->planX), plan->paddedSize[X], HIPFFT_C2C, paddedStorageSize[Z] * paddedSize[Y] / 2) );
//  gpu_safe( hipfftPlan1d(&(plan->invPlanZ), plan->paddedSize[Z], HIPFFT_C2R, 1) );
  gpu_safefft( hipfftPlan1d(&(plan->invPlanZ), plan->paddedSize[Z], HIPFFT_C2R, size[Y]) );
  
  plan->transp = new_gpu_array(plan->paddedStorageN);
  
  return plan;
}


gpuFFT3dPlan* new_gpuFFT3dPlan(int* size){
  return new_gpuFFT3dPlan_padded(size, size); // when size == paddedsize, there is no padding
}

void gpuFFT3dPlan_forward(gpuFFT3dPlan* plan, float* input, float* output){
  //timer_start("gpu_plan3d_real_input_forward_exec");
  
  int* size = plan->size;
  int* pSSize = plan->paddedStorageSize;
  int N0 = pSSize[X];
  int N1 = pSSize[Y];
  int N2 = pSSize[Z]/2; // we treat the complex data as an N0 x N1 x N2 x 2 array
  int N3 = 2;
  
  float* data = input;
  float* data2 = plan->transp; // both the transpose and FFT are out-of-place between data and data2
  
/*	for(int i=0; i<size[X]; i++){
    for(int j=0; j<size[Y]; j++){
      float* rowIn  = &( input[i * pSSize[Y] * pSSize[Z] + j * pSSize[Z]]);
      float* rowOut = &(output[i * pSSize[Y] * pSSize[Z] + j * pSSize[Z]]);
      gpu_safe( hipfftExecR2C(plan->fwPlanZ, (hipfftReal*)rowIn,  (hipfftComplex*)rowOut) );
    }
  }*/
  for(int i=0; i<size[X]; i++){
    float* rowIn  = &( input[i * pSSize[Y] * pSSize[Z]]);
    float* rowOut = &(output[i * pSSize[Y] * pSSize[Z]]);
    gpu_safefft( hipfftExecR2C(plan->fwPlanZ, (hipfftReal*)rowIn,  (hipfftComplex*)rowOut) );
  }
  hipDeviceSynchronize();
  
  gpu_transposeYZ_complex(data, data2, N0, N1, N2*N3);                  // it's now in data2
  
  gpu_safefft( hipfftExecC2C(plan->planY, (hipfftComplex*)data2,  (hipfftComplex*)data2, HIPFFT_FORWARD) ); 
  hipDeviceSynchronize();

  // support for 2D transforms: do not transform if first dimension has size 1
  if(N0 > 1){
    gpu_transposeXZ_complex(data2, data, N0, N2, N1*N3); // size has changed due to previous transpose! // it's now in data2
    gpu_safefft( hipfftExecC2C(plan->planX, (hipfftComplex*)data,  (hipfftComplex*)output, HIPFFT_FORWARD) ); // it's now again in data
    hipDeviceSynchronize();
  }
  else
    memcpy_on_gpu(data2, data, plan->paddedStorageN);             // for N0=1, it's now again in data

  hipDeviceSynchronize();
  //timer_stop("gpu_plan3d_real_input_forward_exec");
}




void gpuFFT3dPlan_inverse(gpuFFT3dPlan* plan, float* input, float* output){
  //timer_start("gpu_plan3d_real_input_inverse_exec");
  
  int* size = plan->size;
  int* pSSize = plan->paddedStorageSize;
  int N0 = pSSize[X];
  int N1 = pSSize[Y];
  int N2 = pSSize[Z]/2; // we treat the complex data as an N0 x N1 x N2 x 2 array
  int N3 = 2;
  
  float* data = input;
  float* data2 = plan->transp; // both the transpose and FFT are out-of-place between data and data2

  if (N0 > 1){
    // input data is XZ transposed and stored in data, FFTs on X-arrays out of place towards data2
    gpu_safefft( hipfftExecC2C(plan->planX, (hipfftComplex*)data,  (hipfftComplex*)data2, HIPFFT_BACKWARD) ); // it's now in data2
    hipDeviceSynchronize();
    gpu_transposeXZ_complex(data2, data, N1, N2, N0*N3); // size has changed due to previous transpose! // it's now in data
  }

	gpu_safefft( hipfftExecC2C(plan->planY, (hipfftComplex*)data,  (hipfftComplex*)data2, HIPFFT_BACKWARD) ); // it's now again in data2
  hipDeviceSynchronize();

  gpu_transposeYZ_complex(data2, data, N0, N2, N1*N3);                 

/*	for(int i=0; i<size[X]; i++){
    for(int j=0; j<size[Y]; j++){
      float* rowIn  = &( input[i * pSSize[Y] * pSSize[Z] + j * pSSize[Z]]);
      float* rowOut = &(output[i * pSSize[Y] * pSSize[Z] + j * pSSize[Z]]);
      gpu_safe( hipfftExecC2R(plan->invPlanZ, (hipfftComplex*)rowIn, (hipfftReal*)rowOut) ); 
    }
  }*/
  for(int i=0; i<size[X]; i++){
    float* rowIn  = &( input[i * pSSize[Y] * pSSize[Z]]);
    float* rowOut = &(output[i * pSSize[Y] * pSSize[Z]]);
    gpu_safefft( hipfftExecC2R(plan->invPlanZ, (hipfftComplex*)rowIn, (hipfftReal*)rowOut) ); 
  }
  hipDeviceSynchronize();
  
  //timer_stop("gpu_plan3d_real_input_inverse_exec");
}


int gpuFFT3dPlan_normalization(gpuFFT3dPlan* plan){
  return plan->paddedSize[X] * plan->paddedSize[Y] * plan->paddedSize[Z];
}

//_____________________________________________________________________________________________ transpose




// //Copied from gpufft by Ben ***********************************************************************
// void gpu_transposeXZ_complex(float* source, float* dest, int N0, int N1, int N2){
//   //timer_start("transposeXZ"); /// @todo section is double-timed with FFT exec
// 
//   if(source != dest){ // must be out-of-place
// 
//   // we treat the complex array as a N0 x N1 x N2 x 2 real array
//   // after transposing it becomes N0 x N2 x N1 x 2
//   N2 /= 2;  ///@todo: should have new variable here!
//   //int N3 = 2;
// 
//   dim3 gridsize(N0, N1, 1); ///@todo generalize!
//   dim3 blocksize(N2, 1, 1);
//   gpu_checkconf(gridsize, blocksize);
//   _gpu_transposeXZ_complex<<<gridsize, blocksize>>>(source, dest, N0, N1, N2);
//   hipDeviceSynchronize();
// 
//   }
// /*  else{
//     gpu_transposeXZ_complex_inplace(source, N0, N1, N2*2); ///@todo see above
//   }*/
//   //timer_stop("transposeXZ");
// }
// 
// __global__ void _gpu_transposeXZ_complex(float* source, float* dest, int N0, int N1, int N2){
//     // N0 <-> N2
//     // i  <-> k
//     int N3 = 2;
// 
//     int i = blockIdx.x;
//     int j = blockIdx.y;
//     int k = threadIdx.x;
// 
//     dest[k*N1*N0*N3 + j*N0*N3 + i*N3 + 0] = source[i*N1*N2*N3 + j*N2*N3 + k*N3 + 0];
//     dest[k*N1*N0*N3 + j*N0*N3 + i*N3 + 1] = source[i*N1*N2*N3 + j*N2*N3 + k*N3 + 1];
// }
// 
// 
// void gpu_transposeYZ_complex(float* source, float* dest, int N0, int N1, int N2){
//   //timer_start("transposeYZ");
// 
//   if(source != dest){ // must be out-of-place
// 
//   // we treat the complex array as a N0 x N1 x N2 x 2 real array
//   // after transposing it becomes N0 x N2 x N1 x 2
//   N2 /= 2;
//   //int N3 = 2;
// 
//   dim3 gridsize(N0, N1, 1); ///@todo generalize!
//   dim3 blocksize(N2, 1, 1);
//   gpu_checkconf(gridsize, blocksize);
//   _gpu_transposeYZ_complex<<<gridsize, blocksize>>>(source, dest, N0, N1, N2);
//   hipDeviceSynchronize();
//   }
// /*  else{
//     gpu_transposeYZ_complex_inplace(source, N0, N1, N2*2); ///@todo see above
//   }*/
//   //timer_stop("transposeYZ");
// }
// 
// __global__ void _gpu_transposeYZ_complex(float* source, float* dest, int N0, int N1, int N2){
//     // N1 <-> N2
//     // j  <-> k
// 
//     int N3 = 2;
// 
//         int i = blockIdx.x;
//     int j = blockIdx.y;
//     int k = threadIdx.x;
// 
// //      int index_dest = i*N2*N1*N3 + k*N1*N3 + j*N3;
// //      int index_source = i*N1*N2*N3 + j*N2*N3 + k*N3;
// 
// 
//     dest[i*N2*N1*N3 + k*N1*N3 + j*N3 + 0] = source[i*N1*N2*N3 + j*N2*N3 + k*N3 + 0];
//     dest[i*N2*N1*N3 + k*N1*N3 + j*N3 + 1] = source[i*N1*N2*N3 + j*N2*N3 + k*N3 + 1];
// /*    dest[index_dest + 0] = source[index_source + 0];
//     dest[index_dest + 1] = source[index_source + 1];*/
// }



// //_____________________________________________________________________________________________ exec plan
// 
// void gpu_plan3d_real_input_forward(gpu_plan3d_real_input* plan, float* data){
//   //timer_start("gpu_plan3d_real_input_forward_exec");
// 
//   int* size = plan->size;
//   int* pSSize = plan->paddedStorageSize;
//   int N0 = pSSize[X];
//   int N1 = pSSize[Y];
//   int N2 = pSSize[Z]/2; // we treat the complex data as an N0 x N1 x N2 x 2 array
//   int N3 = 2;
//   
//   float* data2 = plan->transp; // both the transpose and FFT are out-of-place between data and data2
//   
//   for(int i=0; i<size[X]; i++){
//     for(int j=0; j<size[Y]; j++){
//       float* row = &(data[i * pSSize[Y] * pSSize[Z] + j * pSSize[Z]]);
//       gpu_safe( hipfftExecR2C(plan->fwPlanZ, (hipfftReal*)row,  (hipfftComplex*)row) ); // all stays in data
//     }
//   }
//   hipDeviceSynchronize();
//   
//   gpu_transposeYZ_complex(data, data2, N0, N1, N2*N3);					// it's now in data2
//   gpu_safe( hipfftExecC2C(plan->planY, (hipfftComplex*)data2,  (hipfftComplex*)data2, HIPFFT_FORWARD) ); // it's now again in data
//   hipDeviceSynchronize();
//   
//   gpu_transposeXZ_complex(data2, data, N0, N2, N1*N3); // size has changed due to previous transpose! // it's now in data2
//   gpu_safe( hipfftExecC2C(plan->planX, (hipfftComplex*)data,  (hipfftComplex*)data, HIPFFT_FORWARD) ); // it's now again in data
//   hipDeviceSynchronize();
//   
//   //timer_stop("gpu_plan3d_real_input_forward_exec");
// }
// 
// void gpu_plan3d_real_input_inverse(gpu_plan3d_real_input* plan, float* data){
//   
// }
// 
// void delete_gpu_plan3d_real_input(gpu_plan3d_real_input* plan){
//   
// 	gpu_safe( hipfftDestroy(plan->fwPlanZ) );
// 	gpu_safe( hipfftDestroy(plan->invPlanZ) );
// 	gpu_safe( hipfftDestroy(plan->planY) );
// 	gpu_safe( hipfftDestroy(plan->planX) );
// 
// 	gpu_safe( hipFree(plan->transp) ); 
// 	gpu_safe( hipFree(plan->size) );
// 	gpu_safe( hipFree(plan->paddedSize) );
// 	gpu_safe( hipFree(plan->paddedStorageSize) );
// 	free(plan);
// 
// }


#ifdef __cplusplus
}
#endif