#include "gpu_fft.h"

#include "../macros.h"
#include "gpu_safe.h"
#include "gpu_mem.h"
#include "gpu_transpose.h"

#ifdef __cplusplus
extern "C" {
#endif


/**
 * Creates a new FFT plan for transforming the magnetization. 
 * Zero-padding in each dimension is optional, and rows with
 * only zero's are not transformed.
 * @todo on compute capability < 2.0, the first step is done serially...
 * @todo rename kernelsize -> paddedsize
 */
gpuFFT3dPlan* new_gpuFFT3dPlan_padded(int* size, int* paddedSize){
  
  int N0 = size[X];
  int N1 = size[Y];
  int N2 = size[Z];
  
  assert(paddedSize[X] > 0);
  assert(paddedSize[Y] > 1);
  assert(paddedSize[Z] > 1);
  
  gpuFFT3dPlan* plan = (gpuFFT3dPlan*)malloc(sizeof(gpuFFT3dPlan));
  
  plan->size = (int*)calloc(3, sizeof(int));    ///@todo not int* but int[3]
  plan->paddedSize = (int*)calloc(3, sizeof(int));
  plan->paddedStorageSize = (int*)calloc(3, sizeof(int));
  
//   int* paddedSize = plan->paddedSize;
  int* paddedStorageSize = plan->paddedStorageSize;
  
  plan->size[0] = N0; 
  plan->size[1] = N1; 
  plan->size[2] = N2;
  plan->N = N0 * N1 * N2;
  
  plan->paddedSize[X] = paddedSize[X];
  plan->paddedSize[Y] = paddedSize[Y];
  plan->paddedSize[Z] = paddedSize[Z];
  plan->paddedN = plan->paddedSize[0] * plan->paddedSize[1] * plan->paddedSize[2];
  
  plan->paddedStorageSize[X] = plan->paddedSize[X];
  plan->paddedStorageSize[Y] = plan->paddedSize[Y];
  plan->paddedStorageSize[Z] = gpu_pad_to_stride( plan->paddedSize[Z] + 2 );
  plan->paddedStorageN = paddedStorageSize[X] * paddedStorageSize[Y] * paddedStorageSize[Z];
  
  ///@todo check these sizes !
  gpu_safefft( hipfftPlan1d(&(plan->fwPlanZ), plan->paddedSize[Z], HIPFFT_R2C, 1) );
  gpu_safefft( hipfftPlan1d(&(plan->planY), plan->paddedSize[Y], HIPFFT_C2C, paddedStorageSize[Z] * size[X] / 2) );          // IMPORTANT: the /2 is necessary because the complex transforms have only half the amount of elements (the elements are now complex numbers)
  gpu_safefft( hipfftPlan1d(&(plan->planX), plan->paddedSize[X], HIPFFT_C2C, paddedStorageSize[Z] * paddedSize[Y] / 2) );
  gpu_safefft( hipfftPlan1d(&(plan->invPlanZ), plan->paddedSize[Z], HIPFFT_C2R, 1) );
  
  plan->transp = new_gpu_array(plan->paddedStorageN);
  
  return plan;
}


// gpuFFT3dPlan* new_gpuFFT3dPlan(int* size){
//   return new_gpuFFT3dPlan_padded(size, size); // when size == paddedsize, there is no padding
// }


// void gpuFFT3dPlan_forward(gpuFFT3dPlan* plan, tensor* input, tensor* output){
//   assertDevice(input->list);
//   assertDevice(output->list);
//   assert(input->list == output->list); ///@todo works only in-place for now
//   assert(input->rank == 3);
//   assert(output->rank == 3);
//   for(int i=0; i<3; i++){
//     assert( input->size[i] == plan->paddedStorageSize[i]);
//     assert(output->size[i] == plan->paddedStorageSize[i]);
//   }
//   
//   gpuFFT3dPlan_forward_unsafe(plan, input->list, output->list);
// }


void gpuFFT3dPlan_forward(gpuFFT3dPlan* plan, float* input, float* output){

  int* size = plan->size;
  int* pSSize = plan->paddedStorageSize;
  int N0 = pSSize[X];
  int N1 = pSSize[Y];
  int N2 = pSSize[Z]/2; // we treat the complex data as an N0 x N1 x N2 x 2 array
  int N3 = 2;
  int N = N0*N1*N2*N3;
  float* transp = plan->transp;

  //timer_start("FFT_z");
  for(int i=0; i<size[X]; i++){
    for(int j=0; j<size[Y]; j++){
      float* rowIn  = &( input[i * pSSize[Y] * pSSize[Z] + j * pSSize[Z]]);
      float* rowOut = &(output[i * pSSize[Y] * pSSize[Z] + j * pSSize[Z]]);
      gpu_safefft( hipfftExecR2C(plan->fwPlanZ, (hipfftReal*)rowIn,  (hipfftComplex*)rowOut) );
    }
  }
  hipDeviceSynchronize();
  //timer_stop("FFT_z");

  gpu_transposeYZ_complex(output, transp, N0, N1, N2*N3);
  memcpy_on_gpu(transp, input, N);

  //timer_start("FFT_y");
  gpu_safefft( hipfftExecC2C(plan->planY, (hipfftComplex*)input,  (hipfftComplex*)output, HIPFFT_FORWARD) );
  hipDeviceSynchronize();
  //timer_stop("FFT_y");

  // support for 2D transforms: do not transform if first dimension has size 1
  if(N0 > 1){
    gpu_transposeXZ_complex(output, transp, N0, N2, N1*N3); // size has changed due to previous transpose!
    memcpy_on_gpu(transp, input, N);
    //timer_start("FFT_x");
    gpu_safefft( hipfftExecC2C(plan->planX, (hipfftComplex*)input,  (hipfftComplex*)output, HIPFFT_FORWARD) );
    hipDeviceSynchronize();
    //timer_stop("FFT_x");
  }

}


// void gpuFFT3dPlan_inverse(gpuFFT3dPlan* plan, tensor* input, tensor* output){
//   assertDevice(input->list);
//   assertDevice(output->list);
//   assert(input->list == output->list); ///@todo works only in-place for now
//   assert(input->rank == 3);
//   assert(output->rank == 3);
//   for(int i=0; i<3; i++){
//     assert( input->size[i] == plan->paddedStorageSize[i]);
//     assert(output->size[i] == plan->paddedStorageSize[i]);
//   }
//   gpuFFT3dPlan_inverse_unsafe(plan, input->list, output->list);
// }

void gpuFFT3dPlan_inverse(gpuFFT3dPlan* plan, float* input, float* output){
  
  int* size = plan->size;
  int* pSSize = plan->paddedStorageSize;
  int N0 = pSSize[X];
  int N1 = pSSize[Y];
  int N2 = pSSize[Z]/2; // we treat the complex data as an N0 x N1 x N2 x 2 array
  int N3 = 2;
  int N = N0*N1*N2*N3;
  float* transp = plan->transp;

  if (N0 > 1){
    // input data is XZ transposed
//     timer_start("FFT_x");
    gpu_safefft( hipfftExecC2C(plan->planX, (hipfftComplex*)input,  (hipfftComplex*)output, HIPFFT_BACKWARD) );
    hipDeviceSynchronize();
//     timer_stop("FFT_x");
    gpu_transposeXZ_complex(output, transp, N1, N2, N0*N3); // size has changed due to previous transpose!
    memcpy_on_gpu(transp, input, N);
  }

//   timer_start("FFT_y");
    gpu_safefft( hipfftExecC2C(plan->planY, (hipfftComplex*)input,  (hipfftComplex*)output, HIPFFT_BACKWARD) );
  hipDeviceSynchronize();
//   timer_stop("FFT_y");
  
  gpu_transposeYZ_complex(output, transp, N0, N2, N1*N3);
  memcpy_on_gpu(transp, input, N);

//   timer_start("FFT_z");
    for(int i=0; i<size[X]; i++){
    for(int j=0; j<size[Y]; j++){
      float* rowIn  = &( input[i * pSSize[Y] * pSSize[Z] + j * pSSize[Z]]);
      float* rowOut = &(output[i * pSSize[Y] * pSSize[Z] + j * pSSize[Z]]);
      gpu_safefft( hipfftExecC2R(plan->invPlanZ, (hipfftComplex*)rowIn, (hipfftReal*)rowOut) ); 
    }
  }
  hipDeviceSynchronize();
//   timer_stop("FFT_z");
}


int gpuFFT3dPlan_normalization(gpuFFT3dPlan* plan){
  return plan->paddedSize[X] * plan->paddedSize[Y] * plan->paddedSize[Z];
}



// //_____________________________________________________________________________________________ exec plan
// 
// void gpu_plan3d_real_input_forward(gpu_plan3d_real_input* plan, float* data){
//   timer_start("gpu_plan3d_real_input_forward_exec");
// 
//   int* size = plan->size;
//   int* pSSize = plan->paddedStorageSize;
//   int N0 = pSSize[X];
//   int N1 = pSSize[Y];
//   int N2 = pSSize[Z]/2; // we treat the complex data as an N0 x N1 x N2 x 2 array
//   int N3 = 2;
//   
//   float* data2 = plan->transp; // both the transpose and FFT are out-of-place between data and data2
//   
//   for(int i=0; i<size[X]; i++){
//     for(int j=0; j<size[Y]; j++){
//       float* row = &(data[i * pSSize[Y] * pSSize[Z] + j * pSSize[Z]]);
//       gpu_safe( hipfftExecR2C(plan->fwPlanZ, (hipfftReal*)row,  (hipfftComplex*)row) ); // all stays in data
//     }
//   }
//   hipDeviceSynchronize();
//   
//   gpu_transposeYZ_complex(data, data2, N0, N1, N2*N3);                   // it's now in data2
//   gpu_safe( hipfftExecC2C(plan->planY, (hipfftComplex*)data2,  (hipfftComplex*)data2, HIPFFT_FORWARD) ); // it's now again in data
//   hipDeviceSynchronize();
//   
//   gpu_transposeXZ_complex(data2, data, N0, N2, N1*N3); // size has changed due to previous transpose! // it's now in data2
//   gpu_safe( hipfftExecC2C(plan->planX, (hipfftComplex*)data,  (hipfftComplex*)data, HIPFFT_FORWARD) ); // it's now again in data
//   hipDeviceSynchronize();
//   
//   timer_stop("gpu_plan3d_real_input_forward_exec");
// }
// 
// void gpu_plan3d_real_input_inverse(gpu_plan3d_real_input* plan, float* data){
//   
// }
// 
// void delete_gpu_plan3d_real_input(gpu_plan3d_real_input* plan){
//   
//  gpu_safe( hipfftDestroy(plan->fwPlanZ) );
//  gpu_safe( hipfftDestroy(plan->invPlanZ) );
//  gpu_safe( hipfftDestroy(plan->planY) );
//  gpu_safe( hipfftDestroy(plan->planX) );
// 
//  gpu_safe( hipFree(plan->transp) ); 
//  gpu_safe( hipFree(plan->size) );
//  gpu_safe( hipFree(plan->paddedSize) );
//  gpu_safe( hipFree(plan->paddedStorageSize) );
//  free(plan);
// 
// }



#ifdef __cplusplus
}
#endif
