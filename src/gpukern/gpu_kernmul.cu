#include "hip/hip_runtime.h"
#include "gpu_kernmul.h"
#include "gpu_conf.h"
#include "assert.h"

#ifdef __cplusplus
extern "C" {
#endif


__global__ void _gpu_kernelmul6(float* fftMx,  float* fftMy,  float* fftMz,
                                                     float* fftKxx, float* fftKyy, float* fftKzz,
                                                     float* fftKyz, float* fftKxz, float* fftKxy){
  
  int e = 2 * ((blockIdx.x * blockDim.x) + threadIdx.x);
  
  // we some shared memory here, which saves an "8N" buffer in the global memory
  ///@todo coalescale read/writes
  float reMx = fftMx[e  ];
  float imMx = fftMx[e+1];

  float reMy = fftMy[e  ];
  float imMy = fftMy[e+1];

  float reMz = fftMz[e  ];
  float imMz = fftMz[e+1];

  float Kxx = fftKxx[e];
  float Kyy = fftKyy[e];
  float Kzz = fftKzz[e];

  float Kyz = fftKyz[e];
  float Kxz = fftKxz[e];
  float Kxy = fftKxy[e];
  
  fftMx[e  ] = reMx * Kxx + reMy * Kxy + reMz * Kxz;
  fftMx[e+1] = imMx * Kxx + imMy * Kxy + imMz * Kxz;

  fftMy[e  ] = reMx * Kxy + reMy * Kyy + reMz * Kyz;
  fftMy[e+1] = imMx * Kxy + imMy * Kyy + imMz * Kyz;

  fftMz[e  ] = reMx * Kxz + reMy * Kyz + reMz * Kzz;
  fftMz[e+1] = imMx * Kxz + imMy * Kyz + imMz * Kzz;

}


void gpu_kernelmul6(float* fftMx,  float* fftMy,  float* fftMz,
                                         float* fftKxx, float* fftKyy, float* fftKzz,
                                         float* fftKyz, float* fftKxz, float* fftKxy,
                                         int nRealNumbers){
  
  //timer_start("kernel_mul");
  assert(nRealNumbers > 0);
  assert(nRealNumbers % 2 == 0);
  
   int gridSize = -1;
   int blockSize = -1;
   make1dconf(nRealNumbers/2, &gridSize, &blockSize);

  _gpu_kernelmul6<<<gridSize, blockSize>>>(
                                      fftMx,  fftMy,  fftMz, 
                                      fftKxx, fftKyy, fftKzz,
                                      fftKyz, fftKxz, fftKxy);
  hipDeviceSynchronize();
  //timer_stop("kernel_mul");
}

#ifdef __cplusplus
}
#endif
