#include "hip/hip_runtime.h"
#include "gpu_kernmul.h"
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "assert.h"
#include "timer.h"

#ifdef __cplusplus
extern "C" {
#endif


// |Hx|   |Kxx Kxy Kxz|   |Mx|
// |Hy| = |Kxy Kyy Kyz| * |My|
// |Hz|   |Kxz Kyz Kzz|   |Mz|

__global__ void _gpu_kernelmul6(float* fftMx,  float* fftMy,  float* fftMz,
                                float* fftKxx, float* fftKyy, float* fftKzz,
                                float* fftKyz, float* fftKxz, float* fftKxy, int N){
  int i = threadindex;
  int e = 2 * i;

  // we some shared memory here, which saves an "8N" buffer in the global memory
  ///@todo coalescale read/writes, cleanup indices
  if(i < N){
    float reMx = fftMx[e  ];
    float imMx = fftMx[e+1];

    float reMy = fftMy[e  ];
    float imMy = fftMy[e+1];

    float reMz = fftMz[e  ];
    float imMz = fftMz[e+1];

    float Kxx = fftKxx[i];
    float Kyy = fftKyy[i];
    float Kzz = fftKzz[i];

    float Kyz = fftKyz[i];
    float Kxz = fftKxz[i];
    float Kxy = fftKxy[i];

    fftMx[e  ] = reMx * Kxx + reMy * Kxy + reMz * Kxz;
    fftMx[e+1] = imMx * Kxx + imMy * Kxy + imMz * Kxz;

    fftMy[e  ] = reMx * Kxy + reMy * Kyy + reMz * Kyz;
    fftMy[e+1] = imMx * Kxy + imMy * Kyy + imMz * Kyz;

    fftMz[e  ] = reMx * Kxz + reMy * Kyz + reMz * Kzz;
    fftMz[e+1] = imMx * Kxz + imMy * Kyz + imMz * Kzz;
  }
  
  return;
}

void gpu_kernelmul6(float* fftMx,  float* fftMy,  float* fftMz,
                    float* fftKxx, float* fftKyy, float* fftKzz,
                    float* fftKyz, float* fftKxz, float* fftKxy,
                    int nRealNumbers){

  timer_start("kernel_mul");
  assert(nRealNumbers > 0);
  assert(nRealNumbers % 2 == 0);

  dim3 gridSize, blockSize;
  make1dconf(nRealNumbers/2, &gridSize, &blockSize);

  _gpu_kernelmul6<<<gridSize, blockSize>>>(fftMx,  fftMy,  fftMz,
                                           fftKxx, fftKyy, fftKzz,
                                           fftKyz, fftKxz, fftKxy, nRealNumbers/2);
  gpu_sync();
  timer_stop("kernel_mul");
  
  return;
}




// |Hx|   |Kxx  0   0 |   |Mx|
// |Hy| = | 0  Kyy Kyz| * |My|
// |Hz|   | 0  Kyz Kzz|   |Mz|

__global__ void _gpu_kernelmul4(float* fftMx,  float* fftMy,  float* fftMz,
                                float* fftKxx, float* fftKyy, float* fftKzz, float* fftKyz, int N){
  int i = threadindex;
  int e = 2 * i;

  // we some shared memory here, which saves an "8N" buffer in the global memory
  ///@todo coalescale read/writes, cleanup indices
  if(i < N){
  float reMx = fftMx[e  ];
  float imMx = fftMx[e+1];

  float reMy = fftMy[e  ];
  float imMy = fftMy[e+1];

  float reMz = fftMz[e  ];
  float imMz = fftMz[e+1];

  float Kxx = fftKxx[i];
  float Kyy = fftKyy[i];
  float Kyz = fftKyz[i];
  float Kzz = fftKzz[i];
  
  fftMx[e  ] = reMx * Kxx;
  fftMx[e+1] = imMx * Kxx;
  fftMy[e  ] = reMy * Kyy + reMz * Kyz;
  fftMy[e+1] = imMy * Kyy + imMz * Kyz;
  fftMz[e  ] = reMy * Kyz + reMz * Kzz;
  fftMz[e+1] = imMy * Kyz + imMz * Kzz;
  }
  
  return;
}

void gpu_kernelmul4(float *fftMx, float *fftMy, float *fftMz, 
                    float *fftKxx, float *fftKyy, float *fftKzz, float *fftKyz, 
                    int nRealNumbers){

  timer_start("kernel_mul");
  assert(nRealNumbers > 0);
  assert(nRealNumbers % 2 == 0);

  dim3 gridSize, blockSize;
  make1dconf(nRealNumbers/2, &gridSize, &blockSize);

  _gpu_kernelmul4<<<gridSize, blockSize>>>(fftMx, fftMy, fftMz, fftKxx, fftKyy, fftKzz, fftKyz, nRealNumbers/2);
  gpu_sync();
  timer_stop("kernel_mul");
 
  return;
}



// |Hx|   | 0  0   0 |   |Mx|
// |Hy| = | 0 Kyy Kyz| * |My|
// |Hz|   | 0 Kyz Kzz|   |Mz|

__global__ void _gpu_kernelmul3(float* fftMy,  float* fftMz,
                                float* fftKyy, float* fftKzz, float* fftKyz, int N){
  int i = threadindex;
  int e = 2 * i;

  // we some shared memory here, which saves an "8N" buffer in the global memory
  ///@todo coalescale read/writes, cleanup indices
  if(i < N){

  float reMy = fftMy[e  ];
  float imMy = fftMy[e+1];

  float reMz = fftMz[e  ];
  float imMz = fftMz[e+1];

  float Kyy = fftKyy[i];
  float Kyz = fftKyz[i];
  float Kzz = fftKzz[i];
  
  fftMy[e  ] = reMy * Kyy + reMz * Kyz;
  fftMy[e+1] = imMy * Kyy + imMz * Kyz;
  fftMz[e  ] = reMy * Kyz + reMz * Kzz;
  fftMz[e+1] = imMy * Kyz + imMz * Kzz;
  }
  
  return;
}

void gpu_kernelmul3(float *fftMy, float *fftMz, 
                    float *fftKyy, float *fftKzz, float *fftKyz, 
                    int nRealNumbers){

  timer_start("kernel_mul");
  assert(nRealNumbers > 0);
  assert(nRealNumbers % 2 == 0);

  dim3 gridSize, blockSize;
  make1dconf(nRealNumbers/2, &gridSize, &blockSize);

  _gpu_kernelmul3<<<gridSize, blockSize>>>(fftMy, fftMz, fftKyy, fftKzz, fftKyz, nRealNumbers/2);
  gpu_sync();
  timer_stop("kernel_mul");
 
  return;
}



// |Hx|   | 0   Kz -Ky|   |Jx|
// |Hy| = |-Kz  0   Kx| * |Jy|
// |Hz|   | Ky -Kx  0 |   |Jz|

__global__ void _gpu_kernelmul_biot_savart3D(float* fftJx,  float* fftJy,  float* fftJz,
                                             float* fftKx, float* fftKy, float* fftKz,
                                             int N){
  int i = threadindex;
  int e = 2 * i;

  // we some shared memory here, which saves an "8N" buffer in the global memory
  if(i < N){
    float reJx = fftJx[e  ];
    float imJx = fftJx[e+1];

    float reJy = fftJy[e  ];
    float imJy = fftJy[e+1];

    float reJz = fftJz[e  ];
    float imJz = fftJz[e+1];

    float Kx = fftKx[i];
    float Ky = fftKy[i];
    float Kz = fftKz[i];
    
    fftJx[e  ] =  reJy * Kz - reJz * Ky;
    fftJx[e+1] =  imJy * Kz - imJz * Ky;

    fftJy[e  ] = -reJx * Kz + reJz * Kx;
    fftJy[e+1] = -imJx * Kz + imJz * Kx;

    fftJz[e  ] =  reJx * Ky - reJy * Kx;
    fftJz[e+1] =  imJx * Ky - imJy * Kx;
  }
  
  return;
}

void gpu_kernelmul_biot_savart3D(float* fftJx, float* fftJy, float* fftJz,
                                 float* fftKx, float* fftKy, float* fftKz,
                                 int nRealNumbers){

  timer_start("kernel_mul");
  assert(nRealNumbers > 0);
  assert(nRealNumbers % 2 == 0);

  dim3 gridSize, blockSize;
  make1dconf(nRealNumbers/2, &gridSize, &blockSize);

  _gpu_kernelmul_biot_savart3D<<<gridSize, blockSize>>>(fftJx, fftJy, fftJz,
                                           fftKx, fftKy, fftKz,
                                           nRealNumbers/2);
  gpu_sync();
  timer_stop("kernel_mul");
  
  return;
}



// |Hx|   | 0   Kz -Ky|   |Jx|
// |Hy| = |-Kz  0   0 | * |Jy|
// |Hz|   | Ky  0   0 |   |Jz|

__global__ void _gpu_kernelmul_biot_savart3D_Nx1(float* fftJx, float* fftJy, float* fftJz,
                                                 float* fftKy, float* fftKz,
                                                 int N){
  int i = threadindex;
  int e = 2 * i;

  // we some shared memory here, which saves an "8N" buffer in the global memory
  if(i < N){
    float reJx = fftJx[e  ];
    float imJx = fftJx[e+1];

    float reJy = fftJy[e  ];
    float imJy = fftJy[e+1];

    float reJz = fftJz[e  ];
    float imJz = fftJz[e+1];

    float Ky = fftKy[i];
    float Kz = fftKz[i];
    
    fftJx[e  ] =  reJy * Kz - reJz * Ky;
    fftJx[e+1] =  imJy * Kz - imJz * Ky;

    fftJy[e  ] = -reJx * Kz;
    fftJy[e+1] = -imJx * Kz;

    fftJz[e  ] =  reJx * Ky;
    fftJz[e+1] =  imJx * Ky;
  }
  
  return;
}

void gpu_kernelmul_biot_savart3DNx1(float* fftJx, float* fftJy, float* fftJz,
                                    float* fftKy, float* fftKz,
                                    int nRealNumbers){

  timer_start("kernel_mul");
  assert(nRealNumbers > 0);
  assert(nRealNumbers % 2 == 0);

  dim3 gridSize, blockSize;
  make1dconf(nRealNumbers/2, &gridSize, &blockSize);

  _gpu_kernelmul_biot_savart3D_Nx1<<<gridSize, blockSize>>>(fftJx, fftJy, fftJz, fftKy, fftKz, nRealNumbers/2);
  gpu_sync();
  timer_stop("kernel_mul");
  
  return;
}



// |Hx|   | 0   0  0|   |Jx|
// |Hy| = |-Kz  0  0| * | 0|
// |Hz|   | Ky  0  0|   | 0|

__global__ void _gpu_kernelmul_biot_savart2D(float* fftJx,  float* fftJy,  float* fftJz,
                                             float* fftKy, float* fftKz,
                                             int N){
  int i = threadindex;
  int e = 2 * i;

  // we some shared memory here, which saves an "8N" buffer in the global memory
  if(i < N){
    float reJx = fftJx[e  ];
    float imJx = fftJx[e+1];

    float Ky = fftKy[i];
    float Kz = fftKz[i];
    
    fftJy[e  ] = -reJx * Kz;
    fftJy[e+1] = -imJx * Kz;

    fftJz[e  ] =  reJx * Ky;
    fftJz[e+1] =  imJx * Ky;
  }
  
  return;
}

void gpu_kernelmul_biot_savart2D(float* fftJx,  float* fftJy,  float* fftJz,
                                 float* fftKy, float* fftKz,
                                 int nRealNumbers){

  timer_start("kernel_mul");
  assert(nRealNumbers > 0);
  assert(nRealNumbers % 2 == 0);

  dim3 gridSize, blockSize;
  make1dconf(nRealNumbers/2, &gridSize, &blockSize);

  _gpu_kernelmul_biot_savart2D<<<gridSize, blockSize>>>(fftJx, fftJy, fftJz, fftKy, fftKz, nRealNumbers/2);
  gpu_sync();
  timer_stop("kernel_mul");
  
  return;
}



#ifdef __cplusplus
}
#endif
