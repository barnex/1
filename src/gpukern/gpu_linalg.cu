#include "hip/hip_runtime.h"
#include "gpu_linalg.h"
#include "gpu_safe.h"
#include "gpu_conf.h"

#ifdef __cplusplus
extern "C" {
#endif


///@internal
__global__ void _gpu_add(float* a, float* b, int N){
  int i = threadindex;
  if(i < N){
    a[i] += b[i];
  }
}

void gpu_add(float* a, float* b, int N){
  dim3 gridSize, blockSize;
  make1dconf(N, &gridSize, &blockSize);
  _gpu_add<<<gridSize, blockSize>>>(a, b, N);
  gpu_sync();
}


///@internal
__global__ void _gpu_madd(float* a, float cnst, float* b, int N){
  int i = threadindex;
  if(i < N){
    a[i] += cnst * b[i];
  }
}

void gpu_madd(float* a, float cnst, float* b, int N){
  dim3 gridSize, blockSize;
  make1dconf(N, &gridSize, &blockSize);
  _gpu_madd<<<gridSize, blockSize>>>(a, cnst, b, N);
  gpu_sync();
}


///@internal
__global__ void _gpu_madd2(float* a, float* b, float* c, int N){
  int i = threadindex;
  if(i < N){
    a[i] += b[i] * c[i];
  }
}

void gpu_madd2(float* a, float* b, float* c, int N){
  dim3 gridSize, blockSize;
  make1dconf(N, &gridSize, &blockSize);
  _gpu_madd2<<<gridSize, blockSize>>>(a, b, c, N);
  gpu_sync();
}



///@internal
__global__ void _gpu_add_constant(float* a, float cnst, int N){
  int i = threadindex;
  if(i < N){
    a[i] += cnst;
  }
}

void gpu_add_constant(float* a, float cnst, int N){
  dim3 gridSize, blockSize;
  make1dconf(N, &gridSize, &blockSize);
  _gpu_add_constant<<<gridSize, blockSize>>>(a, cnst, N);
  gpu_sync();
}


///@internal kernel
__global__ void _gpu_linear_combination(float* a, float* b, float weightA, float weightB, int N){
  int i = threadindex;
  if(i < N){
    a[i] = weightA * a[i] + weightB * b[i];
  }
}


void gpu_linear_combination(float* a, float* b, float weightA, float weightB, int N){ 
  dim3 gridSize, blockSize;
  make1dconf(N, &gridSize, &blockSize);
  _gpu_linear_combination<<<gridSize, blockSize>>>(a, b, weightA, weightB, N);
  gpu_sync();
}


///@internal kernel
__global__ void _gpu_linear_combination_many(float* result, float** vectors, float* weights, int NVectors, int NElem){
  int i = threadindex;
  float result_i = result[i];
  
  if(i < NElem){  
    for(int j=0; j<NVectors; j++){
      result_i += weights[j] * vectors[j][i];
    }
    result[i] = result_i;
  }
}


void gpu_linear_combination_many(float* result, float** vectors, float* weights, int NVectors, int NElem){
  dim3 gridSize, blockSize;
  make1dconf(NElem, &gridSize, &blockSize);
  _gpu_linear_combination_many<<<gridSize, blockSize>>>(result, vectors, weights, NVectors, NElem);
  gpu_sync();
}


#ifdef __cplusplus
}
#endif
