#include "hip/hip_runtime.h"
#include "gpu_mem.h"
#include "gpu_safe.h"
#include "gpu_conf.h"
#include "../macros.h"

#ifdef __cplusplus
extern "C" {
#endif

unsigned long long int _gpu_usedmem = 0;


unsigned long long int gpu_usedmem(){
  return _gpu_usedmem;
}


float* new_gpu_array(int size){
  assert(size > 0);
  float* array = NULL;
  gpu_safe( hipMalloc((void**)(&array), size * sizeof(float)) );
  _gpu_usedmem += size * sizeof(float);
  assert(array != NULL); // strange: it seems cuda can return 0 as a valid address?? 
  gpu_zero(array, size);
  return array;
}

void free_gpu_array(float* ptr){
  gpu_safe(hipFree(ptr));
}


float* new_ram_array(int size){
  assert(size > 0);
  float* array = (float*)calloc(size, sizeof(float));
  if(array == NULL){
    fprintf(stderr, "could not allocate %d floats in main memory\n", size);
    abort();
  }
  return array;
}


///@internal kernel
__global__ void _gpu_zero(float* a){
  int i = ((blockIdx.x * blockDim.x) + threadIdx.x);
  a[i] = 0.0f;
}

void gpu_zero(float* data, int nElements){
  debugvv(fprintf(stderr, "gpu_zero(%p, %d)\n", data, nElements));
  gpu_safe( hipMemset(data, 0, nElements*sizeof(float)) );
  gpu_sync();
//   int gridSize = -1, blockSize = -1;
//   make1dconf(nElements, &gridSize, &blockSize);
//   _gpu_zero<<<gridSize, blockSize>>>(data);
//   gpu_sync();
  
}


float* _host_array = NULL;
float* _device_array = NULL;

void assertHost(float* pointer){
  if(_host_array == NULL){
    _host_array = new_ram_array(1);
  }
  _host_array[0] = pointer[0]; // may throw segfault
}

void assertDevice(float* pointer){
  if(_device_array == NULL){
    _device_array = new_gpu_array(1);
  }
  memcpy_on_gpu(pointer, _device_array, 1); // may throw segfault
}

void memcpy_gpu_dir(float* source, float* dest, int nElements, int direction){
  if(direction == 1){
      memcpy_to_gpu(source, dest,nElements);
  }
  else if(direction == 2){
      memcpy_on_gpu(source, dest, nElements);
  }
  else if(direction == 3){
      memcpy_from_gpu(source,  dest, nElements);
  }
  else{
    abort();
  }
}


void memcpy_to_gpu(float* source, float* dest, int nElements){
  assert(nElements > 0);
  int status = hipMemcpy(dest, source, nElements*sizeof(float), hipMemcpyHostToDevice);
  if(status != hipSuccess){
    fprintf(stderr, "CUDA could not copy %d floats from host addres %p to device addres %p\n", nElements, source, dest);
    gpu_safe(status);
  }
  gpu_sync();
}


void memcpy_from_gpu(float* source, float* dest, int nElements){
  debugvv(fprintf(stderr, "memcpy_from_gpu(%p, %p, %d)\n", source, dest, nElements))
  assert(nElements > 0);
  int status = hipMemcpy(dest, source, nElements*sizeof(float), hipMemcpyDeviceToHost);
  if(status != hipSuccess){
    fprintf(stderr, "CUDA could not copy %d floats from device addres %p to host addres %p\n", nElements, source, dest);
    gpu_safe(status);
  }
  gpu_sync(); ///@todo memcpy is synchronous, remove this sync.
}

void memcpy_on_gpu(float* source, float* dest, int nElements){
  assert(nElements > 0);
  int status = hipMemcpy(dest, source, nElements*sizeof(float), hipMemcpyDeviceToDevice);
  if(status != hipSuccess){
    fprintf(stderr, "CUDA could not copy %d floats from device addres %p to device addres %p\n", nElements, source, dest);
    gpu_safe(status);
  }
  gpu_sync();
}

float gpu_array_get(float* dataptr, int index){
  float result = 666.0;
  memcpy_from_gpu(&(dataptr[index]), &result, 1);
  return result;
}


void gpu_array_set(float* dataptr, int index, float value){
  memcpy_to_gpu(&value, &(dataptr[index]), 1);
}


// to avoid having to calculate gpu_stide_float over and over,
// we cache the result of the first invocation and return it
// for all subsequent calls.
// (the function itself is rather expensive)
// -1 means not set yet.

// Arne: FFT strategy:
// It seems a good thing to forget about the gpu stride:
//  * Data sizes are usually quite large powers of 2, so the alignment is already fine.
//  * 1/3 of the FFT's will act on improperly aligned data, but for computation-limited kernels on 2.x GPU's, this should be barely notable
//  * 1/2 of the transposes will be slowed down by about a factor of 2, but R2C FFT's always break the alignment, so there is no workaround anyway.

int _gpu_stride_float_cache = 1;

/* We test for the optimal array stride by creating a 1x1 matrix and checking
 * the stride returned by CUDA.
 */
int gpu_stride_float(){
  if( _gpu_stride_float_cache == -1){
    size_t width = 1;
    size_t height = 1;
    
    float* devPtr;
    size_t pitch;
    gpu_safe( hipMallocPitch((void**)&devPtr, &pitch, width * sizeof(float), height) );
    gpu_safe( hipFree(devPtr) );
    _gpu_stride_float_cache = pitch / sizeof(float);
    debugv( fprintf(stderr, "GPU stride: %d floats\n", _gpu_stride_float_cache) );
  }
  return _gpu_stride_float_cache;
}


void gpu_override_stride(int nFloats){
  assert(nFloats > -2);
  debugv( fprintf(stderr, "GPU stride overridden to %d floats\n", nFloats) );
  _gpu_stride_float_cache = nFloats;
}


int gpu_pad_to_stride(int nFloats){
  assert(nFloats > 0);
  int stride = gpu_stride_float();
  int gpulen = ((nFloats-1)/stride + 1) * stride;
  
  assert(gpulen % stride == 0);
  assert(gpulen > 0);
  assert(gpulen >= nFloats);
  return gpulen;
}


#ifdef __cplusplus
}
#endif
