#include "hip/hip_runtime.h"
#include "gpu_normalize.h"
#include "gpu_conf.h"

#ifdef __cplusplus
extern "C" {
#endif

/// @internal kernel
__global__ void _gpu_normalize_uniform(float* mx , float* my , float* mz){
  int i = ((blockIdx.x * blockDim.x) + threadIdx.x);
  float norm = rsqrtf(mx[i]*mx[i] + my[i]*my[i] + mz[i]*mz[i]);     // inverse square root
  mx[i] *= norm;
  my[i] *= norm;
  mz[i] *= norm;
}

void gpu_normalize_uniform(float* m, int N){

  int gridSize = -1, blockSize = -1;
  make1dconf(N, &gridSize, &blockSize);

  float* mx = &(m[0*N]);
  float* my = &(m[1*N]);
  float* mz = &(m[2*N]);

  _gpu_normalize_uniform<<<gridSize, blockSize>>>(mx, my, mz);
  hipDeviceSynchronize();

}



///@internal kernel
__global__ void _gpu_normalize_map(float* mx , float* my , float* mz, float* normMap){
  int i = ((blockIdx.x * blockDim.x) + threadIdx.x);
  float norm = rsqrtf(mx[i]*mx[i] + my[i]*my[i] + mz[i]*mz[i]) * normMap[i];
  mx[i] *= norm;
  my[i] *= norm;
  mz[i] *= norm;
}

void gpu_normalize_map(float* m, float* map, int N){

  int gridSize = -1, blockSize = -1;
  make1dconf(N, &gridSize, &blockSize);

  float* mx = &(m[0*N]);
  float* my = &(m[1*N]);
  float* mz = &(m[2*N]);

  _gpu_normalize_map<<<gridSize, blockSize>>>(mx, my, mz, map);
  hipDeviceSynchronize();
  
}

#ifdef __cplusplus
}
#endif
