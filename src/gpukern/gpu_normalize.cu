#include "hip/hip_runtime.h"
#include "gpu_normalize.h"
#include "gpu_safe.h"
#include "gpu_conf.h"

#ifdef __cplusplus
extern "C" {
#endif

/// @internal kernel
__global__ void _gpu_normalize_uniform(float* mx , float* my , float* mz, int N){
  int i = threadindex;
  if(i < N){
    float norm = rsqrtf(mx[i]*mx[i] + my[i]*my[i] + mz[i]*mz[i]);     // inverse square root
    mx[i] *= norm;
    my[i] *= norm;
    mz[i] *= norm;
  }
}

void gpu_normalize_uniform(float* m, int N){

  dim3 gridSize, blockSize;
  make1dconf(N, &gridSize, &blockSize);

  float* mx = &(m[0*N]);
  float* my = &(m[1*N]);
  float* mz = &(m[2*N]);

  _gpu_normalize_uniform<<<gridSize, blockSize>>>(mx, my, mz, N);
  gpu_sync();

}



///@internal kernel
__global__ void _gpu_normalize_map(float* mx , float* my , float* mz, float* normMap, int N){
  int i = threadindex;
  if(i < N){
    float norm = rsqrtf(mx[i]*mx[i] + my[i]*my[i] + mz[i]*mz[i]) * normMap[i];
    mx[i] *= norm;
    my[i] *= norm;
    mz[i] *= norm;
  }
}

void gpu_normalize_map(float* m, float* map, int N){

  dim3 gridSize, blockSize;
  make1dconf(N, &gridSize, &blockSize);

  float* mx = &(m[0*N]);
  float* my = &(m[1*N]);
  float* mz = &(m[2*N]);

  _gpu_normalize_map<<<gridSize, blockSize>>>(mx, my, mz, map, N);
  gpu_sync();
  
}

#ifdef __cplusplus
}
#endif
