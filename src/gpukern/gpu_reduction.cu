#include "hip/hip_runtime.h"
// The code in this source file is based on the reduction code from the CUDPP library. Hence the following notice:

/*
Copyright (c) 2007-2010 The Regents of the University of California, Davis
campus ("The Regents") and NVIDIA Corporation ("NVIDIA"). All rights reserved.

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

    * Redistributions of source code must retain the above copyright notice,
      this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright notice,
      this list of conditions and the following disclaimer in the documentation
      and/or other materials provided with the distribution.
    * Neither the name of the The Regents, nor NVIDIA, nor the names of its
      contributors may be used to endorse or promote products derived from this
      software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// This code has been significantly modified from its original version.

#include "gpu_reduction.h"
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "gpu_mem.h"

extern "C"
bool isPow2(unsigned int x){
  return ((x&(x-1))==0);
}

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory {
  __device__ inline operator       T*()
  {
    extern __shared__ int __smem[];
    return (T*)__smem;
  }

  __device__ inline operator const T*() const
  {
    extern __shared__ int __smem[];
    return (T*)__smem;
  }
};

/*
This version adds multiple elements per thread sequentially.  This reduces the overall
cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
(Brent's Theorem optimization)

Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <unsigned int blockSize, bool nIsPow2>
__global__ void _gpu_sum_kernel(float* g_idata, float* g_odata, unsigned int n) {
  float* sdata = SharedMemory<float>();
  
  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
  unsigned int gridSize = blockSize*2*gridDim.x;
  
  float mySum = 0;
  
  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  while (i < n)
  {
    mySum += g_idata[i];
    // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
    if (nIsPow2 || i + blockSize < n)
      mySum += g_idata[i+blockSize];
    i += gridSize;
  }
  
  // each thread puts its local sum into shared memory
  sdata[tid] = mySum;
  __syncthreads();
  
  
  // do reduction in shared mem
  if (blockSize >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
  if (blockSize >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
  if (blockSize >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }

//   #ifndef __DEVICE_EMULATION__
  if (tid < 32)
//     #endif
    {
      // now that we are using warp-synchronous programming (below)
      // we need to declare our shared memory volatile so that the compiler
      // doesn't reorder stores to it and induce incorrect behavior.
      volatile float* smem = sdata;
      if (blockSize >=  64) { smem[tid] = mySum = mySum + smem[tid + 32];  }
      if (blockSize >=  32) { smem[tid] = mySum = mySum + smem[tid + 16];  }
      if (blockSize >=  16) { smem[tid] = mySum = mySum + smem[tid +  8];  }
      if (blockSize >=   8) { smem[tid] = mySum = mySum + smem[tid +  4];  }
      if (blockSize >=   4) { smem[tid] = mySum = mySum + smem[tid +  2];  }
      if (blockSize >=   2) { smem[tid] = mySum = mySum + smem[tid +  1];  }
    }
    // write result for this block to global mem
    if (tid == 0)
      g_odata[blockIdx.x] = sdata[0];
}

#ifdef __cplusplus
extern "C" {
#endif

void gpu_partial_sums(float* d_idata, float* d_odata, int blocks, int threads, int size) {
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

    if (isPow2(size))
    {
      switch (threads)
      {
        case 512: _gpu_sum_kernel<512, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 256: _gpu_sum_kernel<256, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 128: _gpu_sum_kernel<128, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  64: _gpu_sum_kernel< 64, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  32: _gpu_sum_kernel< 32, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  16: _gpu_sum_kernel< 16, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case   8: _gpu_sum_kernel<  8, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case   4: _gpu_sum_kernel<  4, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case   2: _gpu_sum_kernel<  2, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case   1: _gpu_sum_kernel<  1, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
      }
    }
    else
    {
      switch (threads)
      {
        case 512: _gpu_sum_kernel<512, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 256: _gpu_sum_kernel<256, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 128: _gpu_sum_kernel<128, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  64: _gpu_sum_kernel< 64, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  32: _gpu_sum_kernel< 32, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  16: _gpu_sum_kernel< 16, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case   8: _gpu_sum_kernel<  8, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case   4: _gpu_sum_kernel<  4, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case   2: _gpu_sum_kernel<  2, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case   1: _gpu_sum_kernel<  1, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
      }
    }
    gpu_sync();
  }


float gpu_reduce(int operation, float* input, float* dev2, float* host2, int blocks, int threads, int N){
//   fprintf(stderr, "reduce(%d, %p, %p, %p, %d, %d, %d)\n", operation, input, dev2, host2, blocks, threads, N);
  switch(operation){
    default: abort(); break;
    case REDUCE_ADD:
    {
      gpu_partial_sums(input, dev2, blocks, threads, N);
      memcpy_from_gpu(dev2, host2, blocks);
      float sum = 0.;
      for(int i=0; i<blocks; i++){
        sum += host2[i];
      }
//       fprintf(stderr, "sum=%f\n", sum);
      return sum;
    }
  }
}


///@todo leaks memory, should not allocate, for debugging only, use gpu_reduce()
float gpu_sum(float* data, int N){

  assertDevice(data);
  
  int threads = 128;
  while (N <= threads){
    threads /= 2;
  }
  int blocks = divUp(N, threads*2);

  float* dev2 = new_gpu_array(blocks);
  float* host2 = (float*)calloc(blocks, sizeof(float));

  return gpu_reduce(REDUCE_ADD, data, dev2, host2, blocks, threads, N);
}

#ifdef __cplusplus
}
#endif
