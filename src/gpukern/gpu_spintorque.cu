#include "hip/hip_runtime.h"
#include "gpu_spintorque.h"
#include "gpu_safe.h"
#include "../macros.h"

#ifdef __cplusplus
extern "C" {
#endif

#define BLOCKSIZE 16

__global__ void _gpu_directional_diff2D(float ux, float uy, float uz, float* in, float* out, int N0, int N1, int N2, int i){

//int i = i;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int k = blockIdx.y * blockDim.y + threadIdx.y;



}


void gpu_directional_diff2D_async(float ux, float uy, float uz, float *input, float *output, int N0, int N1, int N2, int i){
    dim3 gridsize(divUp(N1, BLOCKSIZE), divUp(N2, BLOCKSIZE));
    dim3 blocksize(BLOCKSIZE, BLOCKSIZE, 1);
    _gpu_directional_diff2D<<<gridsize, blocksize>>>(ux, uy, uz, input, output, N0, N1, N2, i);
}

void gpu_directionial_diff(float ux, float uy, float uz, float* in, float* out, int N0, int N1, int N2){
  for(int i=0; i<N0; i++){
    gpu_directional_diff2D_async(ux, uy, uz, &in[i*N1*N2], &out[i*N1*N2], N0, N1, N2, i);
  }
  gpu_sync();
}


#ifdef __cplusplus
}
#endif
