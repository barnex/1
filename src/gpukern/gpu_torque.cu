#include "hip/hip_runtime.h"
/*
 *  This file is part of MuMax, a high-performance micromagnetic simulator.
 *  Copyright 2010  Arne Vansteenkiste, Ben Van de Wiele.
 *  Use of this source code is governed by the GNU General Public License version 3
 *  (as published by the Free Software Foundation) that can be found in the license.txt file.
 *
 *  Note that you are welcome to modify this code under condition that you do not remove any 
 *  copyright notices and prominently state that you modified it, giving a relevant date.
 */

#include "gpu_torque.h"
#include "gpu_safe.h"
#include "gpu_conf.h"

#ifdef __cplusplus
extern "C" {
#endif

/// @internal
__global__ void _gpu_deltaM(float* mx, float* my, float* mz, float* hx, float* hy, float* hz, float alpha_mul, float* alpha_map, float dt_gilb, int N){

  int i = threadindex;

  if(i < N){
	float alpha;
	if(alpha_map == NULL){
		alpha = alpha_mul;	
	}else{
		alpha = alpha_mul * alpha_map[i];
	}
	
    float Mx = mx[i];
    float My = my[i];
    float Mz = mz[i];
    
    float Hx = hx[i];
    float Hy = hy[i];
    float Hz = hz[i];
    
    //  m cross H
    float _mxHx =  My * Hz - Hy * Mz;
    float _mxHy = -Mx * Hz + Hx * Mz;
    float _mxHz =  Mx * Hy - Hx * My;

    // - m cross (m cross H)
    float _mxmxHx = -My * _mxHz + _mxHy * Mz;
    float _mxmxHy = +Mx * _mxHz - _mxHx * Mz;
    float _mxmxHz = -Mx * _mxHy + _mxHx * My;

    hx[i] = dt_gilb * (_mxHx + _mxmxHx * alpha);
    hy[i] = dt_gilb * (_mxHy + _mxmxHy * alpha);
    hz[i] = dt_gilb * (_mxHz + _mxmxHz * alpha);
  }
}

void gpu_deltaM(float* m, float* h, float alpha_mul, float* alpha_map, float dt_gilb, int N){

  dim3 gridSize, blockSize;
  make1dconf(N, &gridSize, &blockSize);

  float* mx = &(m[0*N]);
  float* my = &(m[1*N]);
  float* mz = &(m[2*N]);

  float* hx = &(h[0*N]);
  float* hy = &(h[1*N]);
  float* hz = &(h[2*N]);

  _gpu_deltaM<<<gridSize, blockSize>>>(mx, my, mz, hx, hy, hz, alpha_mul, alpha_map, dt_gilb, N);
  gpu_sync();
}

#ifdef __cplusplus
}
#endif
