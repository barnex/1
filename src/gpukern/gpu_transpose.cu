#include "hip/hip_runtime.h"
/*
 *  This file is part of MuMax, a high-performance micromagnetic simulator.
 *  Copyright 2010  Arne Vansteenkiste, Ben Van de Wiele.
 *  Use of this source code is governed by the GNU General Public License version 3
 *  (as published by the Free Software Foundation) that can be found in the license.txt file.
 *
 *  Note that you are welcome to modify this code under condition that you do not remove any 
 *  copyright notices and prominently state that you modified it, giving a relevant date.
 */

/**
 * @author Arne
 */
#include "gpu_transpose.h"
#include "gpu_safe.h"
#include "gpu_conf.h"
#include "gpu_stream.h"
#include <assert.h>
#include "timer.h"
#include "../macros.h"

#ifdef __cplusplus
extern "C" {
#endif

typedef struct{
  float real;
  float imag;
}complex;

/// The size of matrix blocks to be loaded into shared memory.
#define BLOCKSIZE 16

///2D
__global__ void _gpu_transpose_complex_YZ(complex* input, complex* output, int N1, int N2, int N)
{
  __shared__ complex block[BLOCKSIZE][BLOCKSIZE+1];

  for (int x=0; x<N; x++){
    // index of the block inside the blockmatrix
    int BI = blockIdx.x;
    int BJ = blockIdx.y;

    // "minor" indices inside the tile
    int i = threadIdx.x;
    int j = threadIdx.y;

    {
      // "major" indices inside the entire matrix
      int I = BI * BLOCKSIZE + i;
      int J = BJ * BLOCKSIZE + j;

      if((I < N1) && (J < N2)){
        block[j][i] = input[x*N1*N2 + J * N1 + I];
      }
    }
    __syncthreads();

    {
      // Major indices with transposed blocks but not transposed minor indices
      int It = BJ * BLOCKSIZE + i;
      int Jt = BI * BLOCKSIZE + j;

      if((It < N2) && (Jt < N1)){
        output[x*N1*N2 + Jt * N2 + It] = block[i][j];
      }
    }
    __syncthreads();
  }
  
  return;
}

/// 2D transpose
void gpu_transpose_complex_YZ(float *input, float *output, int N1, int N2, int N0){
    N2 /= 2;
    dim3 gridsize((N2-1) / BLOCKSIZE + 1, (N1-1) / BLOCKSIZE + 1, 1); // integer division rounded UP. Yes it has to be N2, N1
    dim3 blocksize(BLOCKSIZE, BLOCKSIZE, 1);
    _gpu_transpose_complex_YZ<<<gridsize, blocksize>>>((complex*)input, (complex*)output, N2, N1, N0);
}


__global__ void _gpu_transpose_complex_XZ(complex* input, complex* output, int N1, int N2, int Ny)
{
  __shared__ complex block[BLOCKSIZE][BLOCKSIZE+1];

  for (int y=0; y<Ny; y++){
      // index of the block inside the blockmatrix
    int BI = blockIdx.x;
    int BJ = blockIdx.y;

      // "minor" indices inside the tile
    int i = threadIdx.x;
    int j = threadIdx.y;

      {
        // "major" indices inside the entire matrix
        int I = BI * BLOCKSIZE + i;
        int J = BJ * BLOCKSIZE + j;

        if((I < N1) && (J < N2)){
          block[j][i] = input[J * N1*Ny + y*N1 + I];
        }
      }
      __syncthreads();

      {
        // Major indices with transposed blocks but not transposed minor indices
        int It = BJ * BLOCKSIZE + i;
        int Jt = BI * BLOCKSIZE + j;

        if((It < N2) && (Jt < N1)){
          output[Jt * N2*Ny + y*N2 + It] = block[i][j];
        }
      }
      __syncthreads();
  }
  
  return;
}

void gpu_transpose_complex_XZ(float *input, float *output, int N0, int N1, int N2){
    N2 /= 2;
    dim3 gridsize((N2-1) / BLOCKSIZE + 1, (N0-1) / BLOCKSIZE + 1, 1); // integer division rounded UP. Yes it has to be N2, N0
    dim3 blocksize(BLOCKSIZE, BLOCKSIZE, 1);

    _gpu_transpose_complex_XZ<<<gridsize, blocksize>>>((complex*)input, (complex*)output, N2, N0, N1);

}



///@internal kernel
// __global__ void _gpu_transposeXZ_complex(float* source, float* dest, int N0, int N1, int N2, int j){
//     // N0 <-> N2
//     // i  <-> k
//     int N3 = 2;
// 
//     int i = blockIdx.y * BLOCKSIZE + threadIdx.y;
//     int k = blockIdx.x * BLOCKSIZE + threadIdx.x;
// 
//     if(i < N0 && k < N2){
//       dest[k*N1*N0*N3 + j*N0*N3 + i*N3 + 0] = source[i*N1*N2*N3 + j*N2*N3 + k*N3 + 0];
//       dest[k*N1*N0*N3 + j*N0*N3 + i*N3 + 1] = source[i*N1*N2*N3 + j*N2*N3 + k*N3 + 1];
//     }
// }
// 
// ///@todo this implementation is too slow, especially for "thin" geometries
// void gpu_transposeXZ_complex(float* source, float* dest, int N0, int N1, int N2){
// //    timer_start("transposeXZ");
//   assert(source != dest);{ // must be out-of-place
// 
//   // we treat the complex array as a N0 x N1 x N2 x 2 real array
//   // after transposing it becomes N0 x N2 x N1 x 2
//   N2 /= 2;  ///@todo: should have new variable here!
//   //int N3 = 2;
// 
//   dim3 gridSize(divUp(N2, BLOCKSIZE), divUp(N0, BLOCKSIZE), 1);
//   dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
//   check3dconf(gridSize, blockSize);
// 
//  for(int j=0; j<N1; j++){
//     //_gpu_transposeXZ_complex<<<gridSize, blockSize, gpu_getstream()>>>(source, dest, N0, N1, N2, j);
//     _gpu_transposeXZ_complex<<<gridSize, blockSize>>>(source, dest, N0, N1, N2, j); ///@todo STREAM !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! (x2)
//   }
//   gpu_sync();
// 
//   }
// //  timer_stop("transposeXZ");
// }
// 




/// 2D transpose
// void gpu_transpose_complex_async(float *input, float *output, int N1, int N2){
//     N2 /= 2;
//     dim3 gridsize((N2-1) / BLOCKSIZE + 1, (N1-1) / BLOCKSIZE + 1, 1); // integer division rounded UP. Yes it has to be N2, N1
//     dim3 blocksize(BLOCKSIZE, BLOCKSIZE, 1);
//     _gpu_transpose_complex<<<gridsize, blocksize>>>((complex*)input, (complex*)output, N2, N1);  /// ///////////////// @todo STREAM !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
// }

///@todo need to time this on 2.0 hardware
// void gpu_transposeYZ_complex(float* source, float* dest, int N0, int N1, int N2){
// //   timer_start("transposeYZ");
//   for(int i=0; i<N0; i++){
//     gpu_transpose_complex_async(&source[i*N1*N2], &dest[i*N1*N2], N1, N2);
//   }
//    gpu_sync();
// //    timer_stop("transposeYZ");
// }


#ifdef __cplusplus
}
#endif
